#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "global.h"
#include <assert.h>

#define BLOCK 256
#define RHO0 1E-10
#define P0 1E-10
#define UPWIND
//#define HIO //Activate high-order limiter
#define PRC 1E-16 //Percentage threshold on the high-order limiter
#define LOW_ALPHA //Activate diffusive bound for the high-order limiter
//#define HLLD
//#define PP //Activate Positivity preserving limiter
#define NR 10 //number of Newton-Raphson iterations on the Positivity preserving limiter
//#define CORR

__constant__ double sqrt_mod[5];
__constant__ double sqrts_div[5];
__constant__ double xquad[5];
__constant__ double yquad[5];
__constant__ double wxquad[5];
__constant__ double wyquad[5];
__constant__ double xgll[6];
__constant__ double ygll[6];
__constant__ double wxgll[6];
__constant__ double wygll[6];

__device__  double legendre(double x, int n, int sq){
  double legendre;
  x=min(max(x,-1.0),1.0);
  switch (n) {
  case 0:
    legendre=1.;
    break;
  case 1:
    legendre=x;
    break;
  case 2:
    legendre=0.5*(3.0*x*x-1.0);
    break;
  case 3:
    legendre=(2.5*x*x*x-1.5*x);
    break;
  case 4:
    legendre=0.125*(35.0*x*x*x*x-30.0*x*x+3.0);
    break;
  case 5:
    legendre=0.125*(63.0*pow(x,5)-70.0*pow(x,3)+15.0*x);
    break;
  case 6:
    legendre=1.0/16.0*(231.0*pow(x,6)-315.0*pow(x,4)+105.0*pow(x,2)-5.0);
    break;
  }
  if(sq==1)
    legendre *= sqrt_mod[n];
  return legendre;
}

__device__  double legendre_prime(double x, int n, int sq){
  double legendre_prime;
  x=min(max(x,-1.0),1.0);
  switch (n) {
  case 0:
    legendre_prime=0.0;
    break;
  case 1:
    legendre_prime=1.0;
    break;
  case 2:
    legendre_prime=3.0*x;
    break;
  case 3:
    legendre_prime=0.5*(15.0*x*x-3.0);
    break;
  case 4:
    legendre_prime=0.125*(140.0*x*x*x-60.0*x);
    break;
  case 5:
    legendre_prime=0.125*(315.0*pow(x,4)-210.0*pow(x,2)+15.0);
    break;
  case 6:
    legendre_prime=1.0/16.0*(1386.0*pow(x,5)-1260.0*pow(x,3)+210.0*x);
    break;
  }
  if(sq==1)
    legendre_prime *= sqrt_mod[n];
  return legendre_prime;
}

__device__  double legendre_vector_basis_c(double x, double y, int n, int dim){
  double div_b_basis;
  x=min(max(x,-1.0),1.0);
  y=min(max(y,-1.0),1.0);

  if (dim == 0){
    switch (n){
      case 0:
         div_b_basis = 1.0;
         break;
      case 1:
         div_b_basis = 0.0;
         break;
    }
  }
  else if (dim == 1){
    switch (n){
      case 0:
         div_b_basis = 0.0;
         break;
      case 1:
         div_b_basis= 1.0;
         break;
    }
  }
  return div_b_basis;
}


__device__  double legendre_deriv_vector_basis(double x, double y, int n, int deriv, int dim){
  double div_b_basis;
  x=min(max(x,-1.0),1.0);
  y=min(max(y,-1.0),1.0);


  switch(deriv){
    case 0: // derivative in x
      if (dim==0){
        switch (n){
          case 0:
             div_b_basis = 0.0;
             break;
          case 1:
             div_b_basis = 1.0*sqrt_mod[1];
             break;
          case 2:
             div_b_basis = 0.0;
             break;
          case 3:
             div_b_basis = y*sqrt_mod[1]*sqrt_mod[1];
             break;
          case 4: case 5: case 6: case 7: case 8:
             div_b_basis = 0.0;
             break;
        }
      }
      else if (dim == 1){
        switch (n){
          case 0: case 1: case 2: case 3:
             div_b_basis = 0.0;
             break;
          case 4:
             div_b_basis= 0.0;
             break;
          case 5:
             div_b_basis= 1.0*sqrt_mod[1];
             break;
          case 6:
             div_b_basis= 0.0;//-sqrt(5.)/sqrt(1126.)*45.*y;
             break;
          case 7:
             div_b_basis= y*sqrt_mod[1]*sqrt_mod[1];//-sqrt(3.)/4.*((3.0*sqrt(5.)*y*y-sqrt(5.)-2.0));
             break;
          case 8:
             div_b_basis = 0.0;
             break;
        }
      }
      break;

    case 1: // deriv in y
    if (dim==0){
      switch (n){
        case 0:
           div_b_basis = 0.0;
           break;
        case 1:
           div_b_basis = 0.0;
           break;
        case 2:
           div_b_basis = 1.0*sqrt_mod[1];
           break;
        case 3:
           div_b_basis = x*sqrt_mod[1]*sqrt_mod[1];
           break;
        case 4: case 5: case 6: case 7: case 8:
           div_b_basis = 0.0;
           break;
      }
    }
    else if (dim == 1){
      switch (n){
        case 0: case 1: case 2: case 3:
           div_b_basis = 0.0;
           break;
        case 4:
           div_b_basis= 0.0;
           break;
        case 5:
           div_b_basis= 0.0;
           break;
        case 6:
           div_b_basis= 1.0*sqrt_mod[1];//-sqrt(5.)/sqrt(1126.)*45.*y;
           break;
        case 7:
           div_b_basis= x*sqrt_mod[1]*sqrt_mod[1];//-sqrt(3.)/4.*((3.0*sqrt(5.)*y*y-sqrt(5.)-2.0));
           break;
        case 8:
           div_b_basis = 0.0;
           break;
      }
    }
    break;
        }
  return div_b_basis;
}


// TO DELETE?
__device__  double basis(double x, int n, int sq, int var, int dim){
  double basis;
  x=min(max(x,-1.0),1.0);
  switch (var) {
  case 0: case 1: case 2: case 3: case 6: case 7: case 8:
    basis = legendre(x,n,sq);
    break;
  case 4: //
    //basis = div_y(x,n,sq,dim);//*sqrt(2./dyy);
    basis = legendre(x,n,sq);
    break;
  case 5: //
    basis = legendre(x,n,sq);
    //basis = div_x(x,n,sq,dim);//*sqrt(2./dxx);
    break;
  }
  return basis;
}

__device__  double ldf_div_basis(double x, double y, int n, int dim){
  double div_b_basis;
  x=min(max(x,-1.0),1.0);
  y=min(max(y,-1.0),1.0);

  if (dim==0){
    switch (n){
      case 0:
         div_b_basis = 1.0;
         break;
      case 1:
         div_b_basis = 0.0;
         break;
      case 2:
         div_b_basis = sqrt(3.)/sqrt(2.)*x;
         break;
      case 3:
         div_b_basis = sqrt(3.)*y;
         break;
      case 4:
         div_b_basis =  0.0;
         break;
      // order 3
      case 5:
         div_b_basis = sqrt(30.)*(3.*x*x-1.0)/12.;
         break;
      case 6:
         div_b_basis = -sqrt(30.)*x*y/2.;//1./29.*sqrt(29.)*sqrt(5.)*(3.*x*x-2.);//sqrt(5.)/sqrt(1126.)*0.5*(3.0*x*x-1.0);
         break;
      case 7:
         div_b_basis = sqrt(5.0)*(3.0*y*y-1.0)/2.0;//0.0;//= 0.25*sqrt(6.)*sqrt(5.)*y*(3.0*x*x-1.0);
         break;
      case 8:
         div_b_basis = 0.0;
         break;
      // order 4
      case 9:
          div_b_basis = sqrt(42.0)*sqrt(83.0)*(5.0*x*x*x-4.0*x)/166.0;
          break;
      case 10:
          div_b_basis = sqrt(165585.)*(-56.0*x*x*x/83.0-2.0*x*(12.0*y*y-1.0)+410.0*x/83.0)/1824.0;
          break;
      case 11:
          div_b_basis = sqrt(30.0)*(3.0*x*x*y-1.0*y)/4.0;
          break;
      case 12:
          div_b_basis = sqrt(7.0)*(5.0*y*y*y-3.0*y)/2.0;
          break;
      case 13:
          div_b_basis = 0.0;
          break;
    }
  }
  else if (dim ==1){
    switch (n){
      case 0:
         div_b_basis = 0.0;
         break;
      case 1:
         div_b_basis = 1.0;
         break;
      case 2:
         div_b_basis = -sqrt(3.)/sqrt(2.)*y;
         break;
      case 3:
         div_b_basis=  0.0;
         break;
      case 4:
         div_b_basis=  sqrt(3.)*x;
         break;
      // order 3
      case 5:
         div_b_basis=  -sqrt(30.)*x*y/2.0;//-sqrt(5.)/sqrt(1126.)*0.5*(3.0*y*y-1.0);
         break;
      case 6:
         div_b_basis=  sqrt(30.)*(3.0*y*y-1.0)/12.0;//-sqrt(5.)/sqrt(1126.)*45.*x*y;
         break;
      case 7:
         div_b_basis= 0.0;//= -0.25*sqrt(6.)*sqrt(5.)*x*(3.0*y*y-1.0);
         break;
      case 8:
         div_b_basis = sqrt(5.0)*(3.0*x*x-1.0)/2.0;
         break;
      //order 4
     case 9:
         div_b_basis = sqrt(42.0)*sqrt(83.0)*(-15.*y*x*x+4.0*y)/166.0;
         break;
     case 10:
         div_b_basis = sqrt(165585.)*(8.*y*y*y + 14.*y*(12.*x*x-1.0)/83.0 - 562.*y/83.)/1824.0;
         break;
     case 11:
         div_b_basis = sqrt(30.)*(-3.0*y*y*x + 1.0*x)/4.0;
         break;
     case 12:
         div_b_basis = 0.0 ;
         break;
     case 13:
         div_b_basis = sqrt(7.0)*(5.0*x*x*x-3.0*x)/2.0;
         break;
    }
  }
  return div_b_basis;
}

__device__  double ldf_div_basis_prime(double x, double y, int n, int deriv, int dim){
  double div_b_basis;
  x=min(max(x,-1.0),1.0);
  y=min(max(y,-1.0),1.0);


  switch(deriv){
    case 0: // derivative in x
      if (dim==0){
        switch (n){
          case 0:
             div_b_basis = 0.0;
             break;
          case 1:
             div_b_basis = 0.0;
             break;
          case 2:
             div_b_basis = sqrt(3.)/sqrt(2.);
             break;
          case 3:
             div_b_basis = 0.0;
             break;
          case 4:
             div_b_basis = 0.0;
             break;
          // order 3
          case 5:
             div_b_basis = sqrt(30.)*x/2.;//6./29.*sqrt(29.)*sqrt(5.)*y;//sqrt(5.)/sqrt(1126.)*45.*y;
             break;
          case 6:
             div_b_basis = -sqrt(30.)*y/2.;//sqrt(5.)/sqrt(1126.)*0.5*(6.0*x);
             break;
          case 7:
             div_b_basis = 0.0;// 0.25*sqrt(6.)*sqrt(5.)*y*(6.0*x);//sqrt(3.)/4.*(y*(6.0*sqrt(5.)*x-sqrt(5.)));
             break;
          case 8:
             div_b_basis = 0.0;
             break;
         // order 4
         case 9:
             div_b_basis = sqrt(42.0)*sqrt(83.0)*(15.0*x*x-4.0)/166.0;
             break;
         case 10:
             div_b_basis = sqrt(165585.)*(-168.0*x*x/83.0-2.0*(12.0*y*y-1.0)+410.0/83.0)/1824.0;
             break;
         case 11:
             div_b_basis = sqrt(30.0)*(6.0*x*y)/4.0;
             break;
         case 12:
             div_b_basis = 0.0;
             break;
         case 13:
             div_b_basis = 0.0;
             break;
        }
      }
      else if (dim == 1){
        switch (n){
          case 0:
             div_b_basis = 0.0;
             break;
          case 1:
             div_b_basis = 0.0;
             break;
          case 2:
             div_b_basis = 0.0;
             break;
          case 3:
             div_b_basis= 0.0;
             break;
          case 4:
             div_b_basis= sqrt(3.);
             break;
          // order 3
          case 5:
             div_b_basis=  -sqrt(30.)*y/2.0;
             break;
          case 6:
             div_b_basis=  0.0;
             break;
          case 7:
             div_b_basis= 0.0;
             break;
          case 8:
             div_b_basis = sqrt(5.0)*(6.0*x)/2.0;
             break;
          //order 4
         case 9:
             div_b_basis = sqrt(42.0)*sqrt(83.0)*(-30.*y*x)/166.0;
             break;
         case 10:
             div_b_basis = sqrt(165585.0)*(14.*y*(24.0*x)/83.0)/1824.0;
             break;
         case 11:
             div_b_basis = sqrt(30.)*(-3.0*y*y + 1.0)/4.0;
             break;
         case 12:
             div_b_basis = 0.0 ;
             break;
         case 13:
             div_b_basis = sqrt(7.0)*(15.0*x*x-3.0)/2.0;
             break;
        }
      }
      break;


    case 1: // deriv in y
        if (dim==0){
          switch (n){
            case 0:
               div_b_basis = 0.0;
               break;
            case 1:
               div_b_basis = 0.0;
               break;
            case 2:
               div_b_basis = 0.0;
               break;
            case 3:
               div_b_basis = sqrt(3.);
               break;
            case 4:
               div_b_basis = 0.0;
               break;
            // order 3
            case 5:
               div_b_basis = 0.0;
               break;
            case 6:
               div_b_basis = -sqrt(30.)*x/2.;//1./29.*sqrt(29.)*sqrt(5.)*(3.*x*x-2.);//sqrt(5.)/sqrt(1126.)*0.5*(3.0*x*x-1.0);
               break;
            case 7:
               div_b_basis = sqrt(5.0)*(6.*y)/2.0;//0.0;//= 0.25*sqrt(6.)*sqrt(5.)*y*(3.0*x*x-1.0);
               break;
            case 8:
               div_b_basis = 0.0;
               break;
            // order 4
            case 9:
                div_b_basis = 0.0;
                break;
            case 10:
                div_b_basis = sqrt(165585.0)*(-2.0*x*(24.0*y))/1824.0;
                break;
            case 11:
                div_b_basis = sqrt(30.0)*(3.0*x*x-1.0)/4.0;
                break;
            case 12:
                div_b_basis = sqrt(7.0)*(15.0*y*y-3.0)/2.0;
                break;
            case 13:
                div_b_basis = 0.0;
                break;
          }
        }

        else if (dim ==1){
          switch (n){
            case 0:
               div_b_basis = 0.0;
               break;
            case 1:
               div_b_basis = 0.0;
               break;
            case 2:
               div_b_basis = -sqrt(3.)/sqrt(2.);
               break;
            case 3:
               div_b_basis= 0.0;
               break;
            case 4:
               div_b_basis= 0.0;
               break;
           // order 3
           case 5:
              div_b_basis=  -sqrt(30.)*x/2.0;//-sqrt(5.)/sqrt(1126.)*0.5*(3.0*y*y-1.0);
              break;
           case 6:
              div_b_basis=  sqrt(30.)*y/2.0;//-sqrt(5.)/sqrt(1126.)*45.*x*y;
              break;
           case 7:
              div_b_basis= 0.0;//= -0.25*sqrt(6.)*sqrt(5.)*x*(3.0*y*y-1.0);
              break;
           case 8:
              div_b_basis = 0.0;
              break;
           //order 4
          case 9:
              div_b_basis = sqrt(42.0)*sqrt(83.0)*(-15.*x*x+4.0)/166.0;
              break;
          case 10:
              div_b_basis = sqrt(165585.)*(24.*y*y + 14.*(12.*x*x-1.0)/83.0 - 562./83.)/1824.0;
              break;
          case 11:
              div_b_basis = sqrt(30.)*(-6.0*y*x)/4.0;
              break;
          case 12:
              div_b_basis = 0.0 ;
              break;
          case 13:
              div_b_basis = 0.0;
              break;
          }
        }
          break;
        }
  return div_b_basis;
}


__device__  double basis_ldf(double x, double y, int mx, int my, int k, int var){
  double basis;
  x=min(max(x,-1.0),1.0);
  y=min(max(y,-1.0),1.0);
  switch (var) {
  case 2: case 3:
    if ((mx > k-1) || (my > k-1)){
      basis = 0.0;
      break;
    }
    basis = legendre(x,mx,1)*legendre(y,my,1);
    break;
  case 0: //
    //basis = ldf_div_basis(x,y,mx+my*k,0);
    basis = legendre_vector_basis_c(x,y,mx+my*k,0);
    break;
  case 1: //
    //basis = ldf_div_basis(x,y,mx+k*my,1);
    basis = legendre_vector_basis_c(x,y,mx+k*my,1);
    //basis = div_x(x,n,sq,dim);//*sqrt(2./dxx);
    break;
  }
  return basis;
}


__device__  double basis_ldf_t(double x, double y, int mx, int k, int var){
  double basis;
  int my = 0;
  x=min(max(x,-1.0),1.0);
  y=min(max(y,-1.0),1.0);
  switch (var) {
  case 2: case 3:
    if ((mx > k-1) || (my > k-1)){
      basis = 0.0;
      break;
    }
    //msx = mx
    //msy =
    basis = legendre(x,mx,1)*legendre(y,my,1);
    break;
  case 0: //
    basis = ldf_div_basis(x,y,mx,0);
    break;
  case 1: //
    basis = ldf_div_basis(x,y,mx,1);
    break;
  }
  return basis;
}


__device__  double basis_ldf_prime_t(double x, double y, int mx, int k, int deriv, int var){
  double basis;
  x=min(max(x,-1.0),1.0);
  y=min(max(y,-1.0),1.0);
  switch (var) {
  case 2: case 3:
    if ((mx > k-1) || (mx > k-1)){
      basis = 0.0;
      break;
    }
    basis = legendre(x,mx,1)*legendre(y,mx,1);
    break;
  case 0: //
    basis = ldf_div_basis_prime(x,y,mx,deriv,0);
    break;
  case 1: //
    basis = ldf_div_basis_prime(x,y,mx,deriv,1);
    break;
  }
  return basis;
}


__device__  double minmod(double x, double y, double z){
  double s;
  s=copysign(1.0,x);
  if(copysign(1.0,y) == s && copysign(1.0,z) == s)
    return (double)s*min(fabs(x),min(fabs(y),fabs(z)));
  else
     return 0.0;
}

__device__ int BC(int index, int size, int bc){
  if (bc == 1){//periodic
    if (index == -1)
      index = size-1;
    else if (index == size)
      index = 0;
  }
  else if (bc == 2 || bc == 3 || bc == 4){//transmissive or reflective
    if (index == -1)
      index++;
    else if (index == size)
      index--;
  }
  return index;
}

__device__ double limiting(double* u,int ic,int im,int ip,int jm,int jp,int in,int jn,int m,int b){
  double d_l_x, d_l_y, d_r_x, d_r_y;
  double coeff_i,coeff_j;
  double u_lim;
  int mode;
  u_lim = u[(in+jn*m)*b+ic];
  if(jn > 0){
#ifndef LOW_ALPHA
    coeff_j = sqrt((2.0*double(jn)-1.0)/(2.0*double(jn)+1.0));
#else
    coeff_j = 0.5/sqrt(4.0*double(jn*jn)-1.0);
#endif
    mode = (in+(jn-1)*m)*b;
    d_r_y = coeff_j*(u[mode+jp]-u[mode+ic]);
    d_l_y = coeff_j*(u[mode+ic]-u[mode+jm]);
    u_lim = minmod(u_lim,d_r_y,d_l_y);
  }
  if(in > 0){
#ifndef LOW_ALPHA
    coeff_i = sqrt((2.0*double(in)-1.0)/(2.0*double(in)+1.0));
#else
    coeff_i = 0.5/sqrt(4.0*double(in*in)-1.0);
#endif
    mode = ((in-1)+jn*m)*b;
    d_r_x = coeff_i*(u[mode+ip]-u[mode+ic]);
    d_l_x = coeff_i*(u[mode+ic]-u[mode+im]);
    u_lim = minmod(u_lim,d_r_x,d_l_x);
  }
  return u_lim;
}

__device__ double solve_for_t(double rho, double mx, double my, double e, double rhoa, double mxa, double mya, double ea, double gamma, double eps, int id){
  double a, b, c, d, t, t1, t2;
  int i, iter=NR;
  a = 2.0*(rho-rhoa)*(e-ea) - (mx-mxa)*(mx-mxa) - (my-mya)*(my-mya);
  b = 2.0*(rho-rhoa)*(ea-eps/(gamma-1)) + 2.0*rhoa*(e-ea) - 2.0*(mxa*(mx-mxa)+mya*(my-mya));
  c = 2.0*rhoa*ea - (mxa*mxa+mya*mya) - 2.0*eps*rhoa/(gamma-1.0);
  d = sqrt(fabs(b*b-4.0*a*c));

  if ( (gamma-1.0)*(ea-0.5*(mxa*mxa + mya*mya)/rhoa) < eps){
	t = 0.0;
	return t;
	}
  if ( rhoa < eps){
	t = 0.0;
	return t;
	}

  t1 =1.0 - (a+b+c)/(2*a+b);
  t2 = - c/b;

  if (abs(1.0-t1) > abs(0.0-t2)){
	t = t2;
	}
  else{
	t = t1;
	}

  for(i=0;i<iter-1;i++){
    t = t - (a*t*t+b*t+c)/(2*a*t+b);}

  if (t < 0.0 || t > 1.0 + eps){
	if (c/(a*t) <= 1.0 || c/(a*t) >= 0){
	  t = c/(a*t);}
	else{
	  printf("The other root is not acceptable either.");
	  t = 0.0;
	  }
  }
  return t;
}

__global__ void get_modes_from_nodes(double* nodes, double* modes, int m, int ny, int nx, int nvar){

  int id, ic, jc, im, jm, var;
  int iq, jq, cid;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = nvar*d;
  double val=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  var = id/d;
  ic  = id - var*d;
  jm  = ic/c;
  ic -= jm*c;
  im  = ic/b;
  ic -= im*b;
  jc =  ic/a;
  ic -= jc*a;
  cid = ic + jc*a + var*d;

  if( id < size ){
    for( iq=0; iq < m; iq++){
       for( jq=0; jq < m; jq++)
	 val += 0.25*nodes[iq*b+jq*c+cid]*legendre(xquad[iq],im,1)*legendre(yquad[jq],jm,1)
	   * wxquad[iq]*wyquad[jq];
    }
    modes[id] = val;
  }
}

__global__ void get_nodes_from_modes(double* modes, double* nodes, int m, int ny, int nx, int nvar){

  int id, ic, jc, iq, jq, var;
  int im, jm, cid;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = nvar*d;
  double val=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  var = id/d;
  ic  = id - var*d;
  jq  = ic/c;
  ic -= jq*c;
  iq  = ic/b;
  ic -= iq*b;
  jc =  ic/a;
  ic -= jc*a;
  cid = ic + jc*a + var*d;

  if( id < size ){
    for( im=0; im < m; im++){
       for( jm=0; jm < m; jm++)
	 val += modes[im*b+jm*c+cid]
	   *legendre(xquad[iq],im,1)*legendre(yquad[jq],jm,1);
    }
    nodes[id] = val;
   }
}

__global__ void get_nodes_from_modes_ldf_test_2(double* modes, double* nodes, int m, int ny, int nx, int nvar){

  int id, ic, jc, iq, jq, var;
  int im, cid;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int bsi = m*(m+3)/2;
  int size = d;
  double val1=0.0;
  double val2=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  var = id/d;
  ic  = id - var*d;
  jq  = ic/c;
  ic -= jq*c;
  iq  = ic/b;
  ic -= iq*b;
  jc =  ic/a;
  ic -= jc*a;
  cid = ic + jc*a;

  if( id < size ){
    for( im=0; im < bsi; im++){
        	 val1 += modes[im*b+cid]*basis_ldf_t(xquad[iq],yquad[jq],im,m,0);
           val2 += modes[im*b+cid]*basis_ldf_t(xquad[iq],yquad[jq],im,m,1);
       }
    nodes[id] = val1;
    nodes[id+d] = val2;
    nodes[id+2*d] = 0.0;
    nodes[id+3*d] = 0.0;
   }
}

__global__ void get_modes_from_nodes_ldf_test_2(double* nodes, double* modes, int m, int ny, int nx, int nvar){

  int id, ic, jc, mid;
  int iq, jq, cid, modek;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int bsi = m*(m+3)/2;
  int size = nx*ny*bsi;
  double val=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  modek = id/b;
  ic  = id - modek*b;
  jc =  ic/a;
  ic -= jc*a;
  cid = ic + jc*a;
  mid = modek;

  if( id < size ){
    for( iq=0; iq < m; iq++){
       for( jq=0; jq < m; jq++){
	       val += 0.25*(nodes[iq*b+jq*c+cid]*basis_ldf_t(xquad[iq],yquad[jq],mid,m,0)
	              + nodes[iq*b+jq*c+cid + d]*basis_ldf_t(xquad[iq],yquad[jq],mid,m,1))
    	          * wxquad[iq]*wyquad[jq];
           }
    }
    modes[id] = val;
  }
}

__global__ void compute_primitive(double* u, double* w, double gamma, int usize, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size ){
    w[id] = u[id];
    w[id+usize] = u[id+usize];
    w[id+2*usize] = u[id+2*usize];
    w[id+3*usize] = u[id+3*usize];
  }
}

__global__ void compute_primitive_t(double* u, double* w, double gamma, int m, int usize, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size ){
    w[id] = u[id];
    w[id+usize] = u[id+usize];
  }
}

__global__ void compute_conservative(double* w, double* u, double gamma, int usize, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size ){
    u[id] = w[id];
    u[id+usize]   =  w[id+usize];
    u[id+2*usize]   =  w[id+2*usize];
    u[id+3*usize] = w[id+3*usize];
  }
}

__global__ void cons_to_prim(double* du,double* dw,double gamma,int m,int ny,int nx,int usize,int size){
  int id, cid, ic, jc, mo;
  double rho,vx,vy,vz,drho,dmx,dmy,dmz,dbx,dby,dbz,bx,by,bz,rho_l;
  int a = nx;
  int b = ny*a;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  mo  = id/b;
  ic  = id - mo*b;
  jc  = ic/a;
  ic -= jc*a;
  cid = ic + jc*a;
  if( id < size ){
    id += b;
    drho = dw[id] = du[id];
    rho = dw[cid];
    rho_l = max(rho,rho);
    // central values
    vx = dw[cid+usize];
    vy = dw[cid+usize*2];
    vz = dw[cid+usize*3];

    bx = dw[cid+usize*4];
    by = dw[cid+usize*5];
    bz = dw[cid+usize*6];

    dmx = du[id+usize];
    dmy = du[id+usize*2];
    dmz = du[id+usize*3];

    // velocities
    dw[id+usize]   = (dmx-drho*vx)/rho_l;
    dw[id+usize*2] = (dmy-drho*vy)/rho_l;
    dw[id+usize*3] = (dmz-drho*vz)/rho_l;

    // magnetic field
    dw[id+usize*4] = dbx = du[id+usize*4];
    dw[id+usize*5] = dby = du[id+usize*5];
    dw[id+usize*6] = dbz = du[id+usize*6];

    // pressure
    dw[id+usize*7] = (gamma-1.0)*(du[id+usize*7] + 0.5*drho*(vx*vx+vy*vy+vz*vz)
                                -(vx*dmx+vy*dmy+vz*dmz)
                                -(bx*dbx+by*dby+bz*dbz));

  }
}

__global__ void prim_to_cons(double* dw,double* du,double gamma,int m,int ny,int nx,int usize,int size){
  int id, cid, ic, jc, mo;
  double rho,vx,vy,vz,drho,dvx,dvy,dvz,bx,by,bz,dbx,dby,dbz, rho_l;
  int a = nx;
  int b = ny*a;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  mo = id/b;
  ic  = id - mo*b;
  jc  = ic/a;
  ic -= jc*a;
  cid = ic + jc*a;
  if( id < size ){
    id += b;
    drho = du[id] = dw[id];

    // central values
    rho = dw[cid];
    rho_l = max(rho,rho);
    vx = dw[cid+usize];
    vy = dw[cid+usize*2];
    vz = dw[cid+usize*3];
    bx = dw[cid+usize*4];
    by = dw[cid+usize*5];
    bz = dw[cid+usize*6];

    dvx = dw[id+usize];
    dvy = dw[id+usize*2];
    dvz = dw[id+usize*3];

    dbx = du[id + usize*4] = dw[id + usize*4];
    dby = du[id + usize*5] = dw[id + usize*5];
    dbz = du[id + usize*6] = dw[id + usize*6];

    du[id+usize]   = vx*drho+rho_l*dvx;
    du[id+usize*2] = vy*drho+rho_l*dvy;
    du[id+usize*3] = vz*drho+rho_l*dvz;

    du[id+usize*7] = 0.5*drho*(vx*vx+vy*vy+vz*vz)+rho_l*(vx*dvx+vy*dvy+vz*dvz)
                      + dw[id+usize*7]/(gamma-1.0)
                      + (bx*dbx+by*dby+bz*dbz);
  }
}

__global__ void compute_flux(double* u, double* w, double* flux1, double* flux2, int size){
  int id;
  double psi;
  double b1, b2, b3, v1, v2, v3;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size ){
    b1  = w[id];
    b2  = w[id+size];
    b3  = w[id+2*size];
    v1  = 1.0;
    v2  = 1.0;
    v3  = 0.0;
    psi = w[id+size*3];

    flux1[id] = 0.0;//psi;
    flux1[id+size] = (v1*b2 - v2*b1);
    flux1[id+2*size] = 0.0;

    flux2[id] = (v2*b1-v1*b2);
    flux2[id+size] = 0.0;
    flux2[id+2*size] = 0.0;

    flux1[id+size*3] = 0.0;
    flux2[id+size*3] = 0.0;


   }
}

__global__ void compute_flux_b(double* u, double* w, double* flux1, double* flux2, double ch, int m, int size){
  int id;
  double psi;
  double b1, b2, b3, v1, v2, v3;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size ){
    b1  = w[id];
    b2  = w[id+size];
    b3  = w[id+2*size];
    v1  = 1.0;
    v2  = 1.0;
    v3  = 0.0;
    psi = u[id+size*3];

    flux1[id+size*3] = ch*ch*b1;
    flux2[id+size*3] = ch*ch*b2;
    flux1[id+size] = 0.0;
    flux2[id] = 0.0;
    flux1[id] = psi;
    flux2[id+size] = psi;

   }
}

__global__ void flux_vol (double* f_vol, double* f_q1, double* f_q2, double invdx, double invdy, int m, int ny, int nx, int nvar){
  int id, ic, jc, iq, jq, va;
  int im, jm, cid;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = nvar*d;
  double val1,val2;
  val1=val2=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  va = id/d;
  ic  = id - va*d;
  jm  = ic/c;
  ic -= jm*c;
  im  = ic/b;
  ic -= im*b;
  jc =  ic/a;
  ic -= jc*a;
  cid = ic + jc*a + va*d;
  if( id < size ){
    for( iq=0; iq < m; iq++){
      for( jq=0; jq < m; jq++){
	val1 += f_q1[iq*b+jq*c+cid]*
	  legendre_prime(xquad[iq],im,1)*wxquad[iq]*
	  legendre(yquad[jq],jm,1)*wyquad[jq];
	val2 += f_q2[iq*b+jq*c+cid]*
	  legendre_prime(yquad[jq],jm,1)*wyquad[jq]*
	  legendre(xquad[iq],im,1)*wxquad[iq];
      }
    }
    f_vol[id] = val1*invdx+val2*invdy;
  }
}

__global__ void flux_vol_ldf_t (double* f_vol, double* F, double* G, double invdx, double invdy, int m, int ny, int nx, int nvar){
  int id, ic, jc, iq, jq, va;
  int im, jm, cid, mid;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int modek;
  int bsi = m*(m+3)/2;
  int size = bsi*nx*ny;
  double f1,f2,g1,g2;
  f1=f2=g1=g2=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  modek = id/(b);
  ic  = id - modek*b;
  jc =  ic/a;
  ic -= jc*a;
  cid = ic + jc*a;
  mid = modek;

  if( id < size ){
    for( iq=0; iq < m; iq++){
      for( jq=0; jq < m; jq++){
      	f1 += 0.0;
        g1 += G[iq*b+jq*c+cid]*basis_ldf_prime_t(xquad[iq],yquad[jq],mid,m,1,0)*wxquad[iq]*wyquad[jq];

        f2 += F[iq*b+jq*c+cid+d]*basis_ldf_prime_t(xquad[iq],yquad[jq],mid,m,0,1)*wxquad[iq]*wyquad[jq];
        g2 += 0.0;


      }
    }
    f_vol[id] = f2*invdx + g1*invdy;
  }
}

__global__ void volume_integral (double* f_vol, double* f, double coeff, double invdx, double invdy, int m, int ny, int nx, int nvar){
  int id, ic, jc, iq, jq, va;
  int im, jm, cid;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = nvar*d;
  double val1,val2;
  val1=val2=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  va = id/d;
  ic  = id - va*d;
  jm  = ic/c;
  ic -= jm*c;
  im  = ic/b;
  ic -= im*b;
  jc =  ic/a;
  ic -= jc*a;
  cid = ic + jc*a + va*d;
  if( id < size ){
    for( iq=0; iq < m; iq++){
      for( jq=0; jq < m; jq++){
        //val1 += coeff*f[iq*b+jq*c+cid]*
        //  legendre(xquad[iq],im,1)*wxquad[iq]*
        //  legendre(yquad[jq],jm,1)*wyquad[jq];
        val1 += coeff*f[iq*b+jq*c+cid]*
          legendre(xquad[iq],im,1)*wxquad[iq]*
          legendre(yquad[jq],jm,1)*wyquad[jq];
      }
    }
    f_vol[id] = val1;
  }
}

__global__ void compute_min_dt(double* w, double* Dt, double gamma, double cfl, double dx, double dy, int m, int usize, int size){
  int id, jump;
  double dt,dt_min,csx,csy,constant,cs,d2;
  __shared__ double mins[BLOCK];
  id = threadIdx.x;
  if(id < size){
     constant = cfl/double(2*m-1);

    cs = gamma;
    d2 = 0.5*((w[id]*w[id] + w[id+usize]*w[id+usize]+w[id+usize*2]*w[id+usize*2])/1.0 + cs);
    //csx = sqrt(d2 + sqrt(d2*d2-cs*w[id+usize*3]*w[id+usize*3]/w[id]));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
    //csy = sqrt(d2 + sqrt(d2*d2-cs*w[id+usize*4]*w[id+usize*4]/w[id]));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
    csx = sqrt(d2 + sqrt(d2*d2-cs*1.0));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
    csy = sqrt(d2 + sqrt(d2*d2-cs*1.0));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));

    //dt_min = constant/((fabs(w[id+usize])+csx)/dx + (fabs(w[id+usize*2])+csy)/dy);
    //dt_min = constant/((1.0+csx)/dx + (1.0+csy)/dy);
    dt_min = constant/((1.0)/dx + (1.0)/dy);
    for (id = threadIdx.x+blockDim.x; id < size; id += blockDim.x){ //This is implemented considering only one block in the reduction launch.
      cs = gamma;
      d2 = 0.5*((w[id]*w[id] + w[id+usize]*w[id+usize]+w[id+usize*2]*w[id+usize*2])/1.0 + cs);
      //csx = sqrt(d2 + sqrt(d2*d2-cs*w[id+usize*3]*w[id+usize*3]/w[id]));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
      //csy = sqrt(d2 + sqrt(d2*d2-cs*w[id+usize*4]*w[id+usize*4]/w[id]));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
      csx = sqrt(d2 + sqrt(d2*d2-cs*1.0));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
      csy = sqrt(d2 + sqrt(d2*d2-cs*1.0));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
      //dt = constant/((1.0+csx)/dx + (1.0+csy)/dy);
      dt = constant/((1.0)/dx + (1.0)/dy);

      dt_min=min(dt,dt_min);
    }
    mins[threadIdx.x] = dt_min;
  }
  __syncthreads();
  for(jump = blockDim.x/2; jump > 0; jump >>= 1){
    if( threadIdx.x < jump )
      mins[threadIdx.x]=min(mins[threadIdx.x],mins[threadIdx.x+jump]);
    __syncthreads();
  }
  if(threadIdx.x == 0)
    Dt[0] = mins[0];
}

__global__ void compute_bounds(double* mat, double* bounds, int m, int nx, int ny, int nvar, int size,int var){
  int id,jump;
  int cid, jump2;
  double p,p_max,p_min;
  __shared__ double mins[BLOCK],maxs[BLOCK];
  id = threadIdx.x;
  if(id < size){
    cid = id + size * var;
    p_min = mat[cid];
    p_max = mat[cid];
    for (id = threadIdx.x+blockDim.x; id < size; id += blockDim.x){ //This is implemented considering only one block in the reduction launch.
      p = mat[cid];
      p_min=min(p,p_min);
      p_max=max(p,p_max);
    }
    mins[threadIdx.x] = p_min;
    maxs[threadIdx.x] = p_max;
  }
  __syncthreads();
  for(jump = blockDim.x/2; jump > 0; jump >>= 1){
    if( threadIdx.x < jump ){
      mins[threadIdx.x]=min(mins[threadIdx.x],mins[threadIdx.x+jump]);
      maxs[threadIdx.x]=max(maxs[threadIdx.x],maxs[threadIdx.x+jump]);
     }
    __syncthreads();
  }
  if(threadIdx.x == 0){
    bounds[0] = mins[0];
    bounds[1] = maxs[0];}
}

extern "C" void get_bounds_ (double* mat, int size){
 // size = nx*ny*m*m;
  double bound[2];
  for(int var = 0;var<nvar;var++){
        hipMemset(pivot1,0.0,tsize*sizeof(double));
 	compute_bounds<<<1,BLOCK>>>( mat, pivot1, m, nx, ny, nvar, size, var);
  	hipMemcpy(&bound,pivot1,2*sizeof(double),hipMemcpyDeviceToHost);
  	printf("\n var: %i min: %.14g max: %.14g \n",var, bound[0], bound[1]);
	}
}

__global__ void compute_faces(double* ufaces, double* delta_u, int m, int ny, int nx, int nvar){
  int id, va, ic, jc, im, jm, q, cid, lid, rid, bid, tid;
  double shudl[5],shudr[5],shudb[5],shudt[5];
  double chsi_m = -1, chsi_p = 1, du;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = nvar*b;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  va = id/b;
  ic = id - va*b;
  jc = ic/a;
  ic -= jc*a;
  cid = ic + jc*a + va*d;
  d = 4*c;
  lid = ic + jc*a + va*d;
  rid = lid + c;
  bid = rid + c;
  tid = bid + c;
  if( id < size ){
    for (q=0;q<m;q++)
      shudb[q] = shudt[q] = shudl[q] = shudr[q] = 0.0;
    for (im=0;im<m;im++){
      for (jm=0;jm<m;jm++){
	du = delta_u[im*b+jm*c+cid];
	for (q=0;q<m;q++){
	  shudl[q] += du*legendre(chsi_m,im,1)*legendre(yquad[q],jm,1);
	  shudr[q] += du*legendre(chsi_p,im,1)*legendre(yquad[q],jm,1);
	  shudb[q] += du*legendre(chsi_m,jm,1)*legendre(xquad[q],im,1);
	  shudt[q] += du*legendre(chsi_p,jm,1)*legendre(xquad[q],im,1);
	}
      }
    }
    for (q=0;q<m;q++){
      ufaces[lid+q*b]=shudl[q];
      ufaces[rid+q*b]=shudr[q];
      ufaces[bid+q*b]=shudb[q];
      ufaces[tid+q*b]=shudt[q];
    }
  }
}

__global__ void compute_faces_ldf_t(double* ufaces, double* modesu, int m, int ny, int nx, int nvar){
  int id, va, ic, jc, im, jm, q, cid, lid, rid, bid, tid;
  double shudl1[5],shudr1[5],shudb1[5],shudt1[5];
  double shudl2[5],shudr2[5],shudb2[5],shudt2[5];
  double chsi_m = -1, chsi_p = 1, du;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = b;
  int bsi = m*(m+3)/2;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  ic = id;
  jc = ic/a;
  ic -= jc*a;
  cid = ic + jc*a;// + va*d;
  d = 4*c;
  lid = ic + jc*a;// + va*d;
  rid = lid + c;
  bid = rid + c;
  tid = bid + c;
  if( id < size ){
    for (q=0;q<m;q++){
      shudb1[q] = shudt1[q] = shudl1[q] = shudr1[q] = 0.0;
      shudb2[q] = shudt2[q] = shudl2[q] = shudr2[q] = 0.0;}
    for (im=0;im<  bsi;im++){
      //for (jm=0;jm<m;jm++){
	       du = modesu[im*b+cid];
        	for (q=0;q<m;q++){
        	  shudl1[q] += du*basis_ldf_t(chsi_m,yquad[q],im,m,0);//legendre(chsi_m,im,1)*legendre(yquad[q],jm,1);
        	  shudr1[q] += du*basis_ldf_t(chsi_p,yquad[q],im,m,0);//legendre(chsi_p,im,1)*legendre(yquad[q],jm,1);
        	  shudb1[q] += du*basis_ldf_t(xquad[q],chsi_m,im,m,0);//legendre(chsi_m,jm,1)*legendre(xquad[q],im,1);
        	  shudt1[q] += du*basis_ldf_t(xquad[q],chsi_p,im,m,0);//legendre(chsi_p,jm,1)*legendre(xquad[q],im,1);

            shudl2[q] += du*basis_ldf_t(chsi_m,yquad[q],im,m,1);//legendre(chsi_m,im,1)*legendre(yquad[q],jm,1);
        	  shudr2[q] += du*basis_ldf_t(chsi_p,yquad[q],im,m,1);//legendre(chsi_p,im,1)*legendre(yquad[q],jm,1);
        	  shudb2[q] += du*basis_ldf_t(xquad[q],chsi_m,im,m,1);//legendre(chsi_m,jm,1)*legendre(xquad[q],im,1);
        	  shudt2[q] += du*basis_ldf_t(xquad[q],chsi_p,im,m,1);//legendre(chsi_p,jm,1)*legendre(xquad[q],im,1);
	             }
          //}
      }
    for (q=0;q<m;q++){
      ufaces[lid+q*b]=shudl1[q];
      ufaces[rid+q*b]=shudr1[q];
      ufaces[bid+q*b]=shudb1[q];
      ufaces[tid+q*b]=shudt1[q];

      ufaces[lid+q*b+d]=shudl2[q];
      ufaces[rid+q*b+d]=shudr2[q];
      ufaces[bid+q*b+d]=shudb2[q];
      ufaces[tid+q*b+d]=shudt2[q];

        ufaces[lid+q*b+d*2]=0.0;
        ufaces[rid+q*b+d*2]=0.0;
        ufaces[bid+q*b+d*2]=0.0;
        ufaces[tid+q*b+d*2]=0.0;

        ufaces[lid+q*b+d*3]=0.0;
        ufaces[rid+q*b+d*3]=0.0;
        ufaces[bid+q*b+d*3]=0.0;
        ufaces[tid+q*b+d*3]=0.0;



    }
  }
}

__forceinline__ __device__ double exact_sol(double x, double y, int va, int neql){
	double sol, r;
	switch (neql){
	  case 2:{
	        if (va == 0){
		  sol = exp(-(x+y));}
        	else if (va == 1){
	  	  sol = 0;}
		else if (va == 2){
	  	  sol = 0;}
		else if (va ==3){
	  	  sol = exp(-(x+y));
		}
    	  	break;}
	  case 17:{
		double rho_0, r0, n, grav, Ms, eps, h, vk, cs, vt, omega;
    		rho_0 = 1;
    		r0 = 0.275*10.;
    		n = 10;
    		grav = 1;
    		Ms = 1;
    		eps = 1E-1;
    		h = 0.03;
            	r = sqrt(pow(x,2.) + pow(y,2.));
            	omega = sqrt(1./pow((r*r + eps*eps),3./2.));
            	vk = sqrt(r/pow((r*r+eps*eps),3./2.));
            	cs = h*vk;
            	vt = sqrt(-(-2*h*h*r/pow(eps*eps+r*r,3./2.)+3*h*h/(eps*eps+r*r))*r+(r*r)/pow((eps*eps+r*r),3./2.));
 	        if (va == 0){sol = rho_0/(1.+(r/r0)*(r/r0));}
        	else if (va == 1){sol = -vt/r*y;}
		else if (va == 2){sol = vt/r*x;}
		else if (va ==3){sol = cs*cs*rho_0/(1.+(r/r0)*(r/r0));}
		break;}
	  case 19:{
	        if (va == 0){
		  sol = exp(-(x));}
        	else if (va == 1){
	  	  sol = 0;}
		else if (va == 2){
	  	  sol = 0;}
		else if (va ==3){
	  	  sol = exp(-(x));
		}
    	  	break;}
	  default:{
	    	break;}
  	}

	return sol;
}

__global__ void compute_GxGLL(double* nodesX, double* nodesY, double* modes, int k, int m, int ny, int nx, int nvar){
  int id, va, ic, jc, gl, gll, iq, jq, cid;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = k*c;
  int size = d*nvar;
  double valx,valy,u;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  va = id/d;
  ic  = id - va*d;
  gll = ic/c;
  ic -= gll*c;
  gl  = ic/b;
  ic -= gl*b;
  jc  = ic/a;
  ic -= jc*a;

  c = m*b;
  d = m*c;
  cid = ic + jc*a + va*d;

  if( id < size ){
    valx=valy=0.0;
    for(iq=0;iq<m;iq++){
      for(jq=0;jq<m;jq++){
	u=modes[iq*b+jq*c+cid];
	valx += u*legendre(xgll[gll],iq,1)*legendre(yquad[gl],jq,1);
	valy += u*legendre(ygll[gll],jq,1)*legendre(xquad[gl],iq,1);
      }
    }
    nodesX[id]=valx;
    nodesY[id]=valy;
  }
}

__global__ void compute_LLF(double* u, double* w, double* f, double* FG,
			   double gamma, int m, int ny, int nx, int nvar, int dim, int bc, int size){
  int id, cid, var, cell, face, quad, mc, pc, pid, mid, fsize;
  double speed_m, speed_p, cmax;
  double bnormp, bnormm, c2p, c2m, d2p, d2m;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  int a = nx+1-dim;
  int b = (ny+dim)*a;
  int c = m*b;
  int d;
  quad = id/b;
  if(dim == 0){
    face = id-quad*b;
    cell = face/a;
    face -= cell*a;
    cid = cell*nx + quad*nx*ny;
    fsize = nx;
    a = 1;
  }
  else if(dim == 1){
    cell = id-quad*b;
    face = cell/a;
    cell -= face*a;
    cid = cell + quad*nx*ny;
    fsize = ny;
    a = nx;
  }
  b=m*nx*ny*(2*dim);
  c=m*nx*ny*(2*dim+1);
  d=4*m*nx*ny;
  if(id < size){
    mc = BC(face-1,fsize,bc);
    pc = BC(face,fsize,bc);
    mid = cid+mc*a+c;
    pid = cid+pc*a+b;

    bnormp = w[pid+d*5]*w[pid+d*5] + w[pid+d*4]*w[pid+d*4]+w[pid+d*6]*w[pid+d*6];
    bnormm = w[mid+d*5]*w[mid+d*5] + w[mid+d*4]*w[mid+d*4]+w[mid+d*6]*w[mid+d*6];
    c2p = gamma*w[pid+d*7]/w[pid];
    c2m = gamma*w[mid+d*7]/w[mid];
    d2p = 0.5*(bnormp/w[pid] + c2p);
    d2m = 0.5*(bnormm/w[mid] + c2m);

    speed_p = fabs(w[pid+d*(dim+1)])+ sqrt(d2p + sqrt(d2p*d2p-c2p*w[pid+d*(4+dim)]*w[pid+d*(4+dim)]/w[pid])); //sqrt(gamma*max(w[pid+d*3],P0)/max(w[pid],RHO0)));
    speed_m = fabs(w[mid+d*(dim+1)])+ sqrt(d2m + sqrt(d2m*d2m-c2m*w[mid+d*(4+dim)]*w[mid+d*(4+dim)]/w[mid]));//sqrt(gamma*max(w[mid+d*3],P0)/max(w[mid],RHO0)));

    cmax=max(speed_m,speed_p);
    for(var = 0; var < nvar; var++){
      FG[id+size*var]=0.5*(f[pid+d*var]+f[mid+d*var])-0.5*cmax*(u[pid+d*var]-u[mid+d*var]);
    }
  }
}

__global__ void compute_upwind(double* u, double* w, double* f, double* FG,
			   double gamma, int m, int ny, int nx, int nvar, int dim, int bc, int size){
  int id, cid, var, cell, face, quad, mc, pc, pid, mid, fsize;
  double speed_m, speed_p, cmax;
  double bnormp, bnormm, c2p, c2m, d2p, d2m;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  int a = nx+1-dim;
  int b = (ny+dim)*a;
  int c = m*b;
  int d;
  quad = id/b;
  if(dim == 0){
    face = id-quad*b;
    cell = face/a;
    face -= cell*a;
    cid = cell*nx + quad*nx*ny;
    fsize = nx;
    a = 1;
  }
  else if(dim == 1){
    cell = id-quad*b;
    face = cell/a;
    cell -= face*a;
    cid = cell + quad*nx*ny;
    fsize = ny;
    a = nx;
  }
  b=m*nx*ny*(2*dim);
  c=m*nx*ny*(2*dim+1);
  d=4*m*nx*ny;
  if(id < size){
    mc = BC(face-1,fsize,bc);
    pc = BC(face,fsize,bc);
    mid = cid+mc*a+c;
    pid = cid+pc*a+b;

    bnormp = w[pid]*w[pid] + w[pid+d]*w[pid+d];//+w[pid+d*2]*w[pid+d*2];
    bnormm = w[mid]*w[mid] + w[mid+d]*w[mid+d];//+w[mid+d*2]*w[mid+d*2];
    c2p = gamma;
    c2m = gamma;
    d2p = 0.5*(bnormp + c2p);
    d2m = 0.5*(bnormm + c2m);

    speed_p = fabs(1.0);// + sqrt(d2p + sqrt(d2p*d2p-c2p)); //sqrt(gamma*max(w[pid+d*3],P0)/max(w[pid],RHO0)));
    speed_m = fabs(1.0);// + sqrt(d2m + sqrt(d2m*d2m-c2m));//sqrt(gamma*max(w[mid+d*3],P0)/max(w[mid],RHO0)));

    cmax=max(speed_m,speed_p);

    for(var = 0; var < nvar; var++){

	  FG[id+size*var]=0.5*(f[pid+d*var]+f[mid+d*var])-0.5*cmax*(u[pid+d*var]-u[mid+d*var]);
    //FG[id+size*var]=f[mid+d*var];

    }
  }
}

__global__ void compute_true_upwind(double* u, double* w, double* f, double* FG,
			   double gamma, int m, int ny, int nx, int nvar, int dim, int bc, int size){
  int id, cid, var, cell, face, quad, mc, pc, pid, mid, fsize;
  double speed_m, speed_p, cmax;
  double bnormp, bnormm, c2p, c2m, d2p, d2m;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  int a = nx+1-dim;
  int b = (ny+dim)*a;
  int c = m*b;
  int d;
  quad = id/b;
  if(dim == 0){
    face = id-quad*b;
    cell = face/a;
    face -= cell*a;
    cid = cell*nx + quad*nx*ny;
    fsize = nx;
    a = 1;
  }
  else if(dim == 1){
    cell = id-quad*b;
    face = cell/a;
    cell -= face*a;
    cid = cell + quad*nx*ny;
    fsize = ny;
    a = nx;
  }
  b=m*nx*ny*(2*dim);
  c=m*nx*ny*(2*dim+1);
  d=4*m*nx*ny;
  if(id < size){
    mc = BC(face-1,fsize,bc);
    pc = BC(face,fsize,bc);
    mid = cid+mc*a+c;
    pid = cid+pc*a+b;

    bnormp = w[pid]*w[pid] + w[pid+d]*w[pid+d];//+w[pid+d*2]*w[pid+d*2];
    bnormm = w[mid]*w[mid] + w[mid+d]*w[mid+d];//+w[mid+d*2]*w[mid+d*2];
    c2p = gamma;
    c2m = gamma;
    d2p = 0.5*(bnormp + c2p);
    d2m = 0.5*(bnormm + c2m);

    speed_p = fabs(1.0);// + sqrt(d2p + sqrt(d2p*d2p-c2p)); //sqrt(gamma*max(w[pid+d*3],P0)/max(w[pid],RHO0)));
    speed_m = fabs(1.0);// + sqrt(d2m + sqrt(d2m*d2m-c2m));//sqrt(gamma*max(w[mid+d*3],P0)/max(w[mid],RHO0)));

    cmax=max(speed_m,speed_p);

    for(var = 0; var < nvar; var++){

      // if in Bx --
      if ((var == 0)&&(dim==0)){
        //FG[id+size*var]=f[mid+d*var];
        FG[id+size*var]=0.5*(f[pid+d*var]+f[mid+d*var]); //-0.5*cmax*(u[pid+d*var]-u[mid+d*var]);
      }
      if ((var == 0)&&(dim==1)){
        //FG[id+size*var]=0.5*(f[pid+d*var]+f[mid+d*var]);
        FG[id+size*var]=0.5*(f[pid+d*var]+f[mid+d*var]) - 0.5*(u[pid+d*var]-u[mid+d*var]) + 0.5*(u[pid+d*1]+u[mid+d*1]);
      }
      // if in By
      if ((var == 1)&&(dim==0)){
        FG[id+size*var]=0.5*(f[pid+d*var]+f[mid+d*var]) + 0.5*(u[pid+d*0]-u[mid+d*0]) - 0.5*(u[pid+d*var]+u[mid+d*var]);
        //FG[id+size*var]=0.5*(f[pid+d*var]+f[mid+d*var]);
      }
      if ((var == 1)&&(dim==1)){
        //FG[id+size*var]=0.5*(f[pid+d*var]+f[mid+d*var])-0.5*cmax*(u[pid+d*var]-u[mid+d*var]);
        FG[id+size*var]=0.5*(f[pid+d*var]+f[mid+d*var]);
      }

    }
  }
}

__global__ void compute_average(double* u, double* w, double* f, double* FG,
			   double gamma, int m, int ny, int nx, int nvar, int dim, int bc, int size){
  int id, cid, var, cell, face, quad, mc, pc, pid, mid, fsize;
  double speed_m, speed_p, cmax;
  double bnormp, bnormm, c2p, c2m, d2p, d2m;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  int a = nx+1-dim;
  int b = (ny+dim)*a;
  int c = m*b;
  int d;
  quad = id/b;
  if(dim == 0){
    face = id-quad*b;
    cell = face/a;
    face -= cell*a;
    cid = cell*nx + quad*nx*ny;
    fsize = nx;
    a = 1;
  }
  else if(dim == 1){
    cell = id-quad*b;
    face = cell/a;
    cell -= face*a;
    cid = cell + quad*nx*ny;
    fsize = ny;
    a = nx;
  }
  b=m*nx*ny*(2*dim);
  c=m*nx*ny*(2*dim+1);
  d=4*m*nx*ny;
  if(id < size){
    mc = BC(face-1,fsize,bc);
    pc = BC(face,fsize,bc);
    mid = cid+mc*a+c;
    pid = cid+pc*a+b;

    bnormp = w[pid]*w[pid] + w[pid+d]*w[pid+d];//+w[pid+d*2]*w[pid+d*2];
    bnormm = w[mid]*w[mid] + w[mid+d]*w[mid+d];//+w[mid+d*2]*w[mid+d*2];
    c2p = gamma;
    c2m = gamma;
    d2p = 0.5*(bnormp + c2p);
    d2m = 0.5*(bnormm + c2m);

    speed_p = fabs(1.0) + sqrt(d2p + sqrt(d2p*d2p-c2p)); //sqrt(gamma*max(w[pid+d*3],P0)/max(w[pid],RHO0)));
    speed_m = fabs(1.0) + sqrt(d2m + sqrt(d2m*d2m-c2m));//sqrt(gamma*max(w[mid+d*3],P0)/max(w[mid],RHO0)));

    cmax=max(speed_m,speed_p);

    for(var = 0; var < nvar; var++){

	  FG[id+size*var]= 0.0; //0.5*(f[pid+d*var]+f[mid+d*var])-0.5*cmax*(u[pid+d*var]-u[mid+d*var]);
    //FG[id+size*var]=f[mid+d*var];

    }
  }
}


__global__ void compute_HLLC(double* u,double* w,double* FG,double gamma,int m,int ny,int nx,int nvar,int dim,int bc,int size){

  int id, cid, cell, quad, mc, pc, pid, mid, face, fsize;
  int dim1, dim2;
  double cmax, cp, cm, vp, vm, sp, sm, dp, dm, pp, pm, vstar, e;
  double wgdnv[4];
  id = blockDim.x * blockIdx.x + threadIdx.x;
  int a = nx+1-dim;
  int b = (ny+dim)*a;
  int c = m*b;
  int d;
  quad = id/b;
  if(dim == 0){
    face = id-quad*b;
    cell = face/a;
    face -= cell*a;
    cid = cell*nx + quad*nx*ny;
    fsize = nx;
    a = 1;
  }
  else if(dim == 1){
    cell = id-quad*b;
    face = cell/a;
    cell -= face*a;
    cid = cell + quad*nx*ny;
    fsize = ny;
    a = nx;
  }
  b=m*nx*ny*(2*dim); //Index for left/bottom face
  c=m*nx*ny*(2*dim+1); //Index for right/top face
  d=4*m*nx*ny;
  if(id < size){
    dim1 = dim+1;
    dim2 = 2-dim;
    mc = BC(face-1,fsize,bc);
    pc = BC(face,fsize,bc);
    pid = cid+pc*a+b; //UR=UL(face)
    mid = cid+mc*a+c; //UL=UR(face-1)
    cp = sqrt(gamma*max(w[pid+d*3],P0)/max(w[pid],RHO0));
    cm = sqrt(gamma*max(w[mid+d*3],P0)/max(w[mid],RHO0));
    cmax=max(cm,cp);
    vp = w[pid+d*dim1];
    vm = w[mid+d*dim1];
    pp = w[pid+d*3];
    pm = w[mid+d*3];
    //Compute HLL wave speed
    sm=min(vp,vm)-cmax;
    sp=max(vp,vm)+cmax;
    //Compute Lagrangian sound speed
    dm=w[mid]*(vm-sm);
    dp=w[pid]*(sp-vp);
    //Compute acoustic star state
    vstar=(dp*vp+dm*vm+(pm-pp))/(dm+dp);
    if(sm>0.0){
      wgdnv[0]=w[mid];
      wgdnv[dim1]=vm;
      wgdnv[dim2]=w[mid+d*dim2];
      wgdnv[3]=pm;
      e=u[mid+d*3];
    }
    else if(vstar>0.0){
      wgdnv[0]=w[mid]*(sm-vm)/(sm-vstar);;
      wgdnv[dim1]=vstar;
      wgdnv[dim2]=w[mid+d*dim2];
      wgdnv[3]=w[mid+d*3]+w[mid]*(sm-vm)*(vstar-vm);
      e=((sm-vm)*u[mid+d*3]-pm*vm+wgdnv[3]*vstar)/(sm-vstar);
    }
    else if(sp>0.0){
      wgdnv[0]=w[pid]*(sp-vp)/(sp-vstar);
      wgdnv[dim1]=vstar;
      wgdnv[dim2]=w[pid+d*dim2];
      wgdnv[3]=w[pid+d*3]+w[pid]*(sp-vp)*(vstar-vp);
      e=((sp-vp)*u[pid+d*3]-pp*vp+wgdnv[3]*vstar)/(sp-vstar);
    }
    else{
      wgdnv[0]=w[pid];
      wgdnv[dim1]=vp;
      wgdnv[dim2]=w[pid+d*dim2];
      wgdnv[3]=pp;
      e=u[pid+d*3];
    }
    FG[id]=wgdnv[0]*wgdnv[dim1];
    FG[id+size*dim1]=wgdnv[0]*wgdnv[dim1]*wgdnv[dim1]+wgdnv[3];
    FG[id+size*dim2]=wgdnv[0]*wgdnv[dim1]*wgdnv[dim2];
    FG[id+size*3]=wgdnv[dim1]*(e+wgdnv[3]);
  }
}

__global__ void compute_HLLD(double* u,double* w,double* FG,double gamma,int m,int ny,int nx,int nvar,int dim,int bc,int size){
  int id, cid, cell, quad, mc, pc, pid, mid, face, fsize;
  int dim1, dim2, bdim1, bdim2;
  double cmax, cp, cm, vp, vm, sp, sm, dp, dm, pp, pm, e, um, up, Ptotm, Ptotp;
  double wgdnv[8];
  double bnormal, bp, bm, rm, rp, Pm, Pp, etoto, wm, wp, Bt1m, Bt2m, Bt1p, Bt2p, sg;
  double Emagm, Emagp, Etotm, Etotp, c2m, c2p, d2m, d2p, SM, SP, cfastm, cfastp, ustar;
  double vstarm, vstarp, vstarstar, Ptotstar, Bt1starm, Bt1starp, Bt2starm, Bt2starp, Bt2starstar, estarstarm, estarstarp, estarm, estarp, SAM, SAP;
  double rstarm, rstarp, wstarp, wstarstar, estar,el, vdotBm, vdotBp, wstarm, Bt1starstar, vdotb, bnorm;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  int a = nx+1-dim;
  int b = (ny+dim)*a;
  int c = m*b;
  int d;
  quad = id/b;
  if(dim == 0){ //x normal
    face = id-quad*b;
    cell = face/a;
    face -= cell*a;
    cid = cell*nx + quad*nx*ny;
    fsize = nx;
    a = 1;
  }
  else if(dim == 1){ //y normal
    cell = id-quad*b;
    face = cell/a;
    cell -= face*a;
    cid = cell + quad*nx*ny;
    fsize = ny;
    a = nx;
  }
  b=m*nx*ny*(2*dim); //Index for left/bottom face
  c=m*nx*ny*(2*dim+1); //Index for right/top face
  d=4*m*nx*ny;
  if(id < size){
    dim1 = dim+1;
    dim2 = 2-dim;

    //bdim1 = dim1+3; // if we are in x-direction, bdim1 = 4, bdim2 = 5
    //bdim2 = dim2+3; // if we are in y-direction, bdim1 = 5, bdim2 = 4

    mc = BC(face-1,fsize,bc);
    pc = BC(face,fsize,bc);
    pid = cid+pc*a+b; //UR=UL(face)
    mid = cid+mc*a+c; //UL=UR(face-1)

    // continuity on normal face
    bnormal = 0.5*(w[pid+d*(dim1+3)]+w[mid+d*(dim1+3)]); //either 4 or 5
    sg = copysign(1.0,bnormal);

    // left variables
    rm = w[mid];
    Pm = w[mid + d*7];
    um = w[mid + d*dim1];
    vm = w[mid + d*dim2];
    wm = w[mid + d*3];
    bm = bnormal;
    Bt1m = w[mid + d*(dim2+3)];
    Bt2m = w[mid + d*6];
    Emagm = 0.5*(bm*bm + Bt1m*Bt1m + Bt2m*Bt2m);
    Etotm = Pm*1./(gamma-1) + 0.5*(um*um + vm*vm + wm*wm)*rm + Emagm;
    Ptotm = Pm + Emagm;

    // right variables
    rp = w[pid];
    Pp = w[pid + d*7];
    up = w[pid + d*dim1];
    vp = w[pid + d*dim2];
    wp = w[pid + d*3];
    bp = bnormal;
    Bt1p = w[pid + d*(dim2+3)];
    Bt2p = w[pid + d*6];
    Emagp = 0.5*(bp*bp + Bt1p*Bt1p + Bt2p*Bt2p);
    Etotp = Pp*1./(gamma-1.) + 0.5*(up*up + vp*vp + wp*wp)*rp + Emagp;
    Ptotp  = Pp + Emagp;

    // Find the largest eigenvalues in the normal direction to the interface
    c2m = gamma*Pm/rm;
    d2m = Emagm/rm + 0.5*c2m;
    cfastm = sqrt( d2m + sqrt(d2m*d2m - c2m*bm*bm/rm));

    c2p = gamma*Pp/rp;
    d2p = Emagp/rp + 0.5*c2p;
    cfastp = sqrt( d2p + sqrt(d2p*d2p - c2p*bp*bp/rp));

    // Compute HLL wave speed
    SM = min(um,up)-max(cfastp,cfastm); //different from paper ?
    SP = max(um,up)+max(cfastp,cfastm);

    //Compute Lagrangian sound speed
    dm = rm*(um-SM);
    dp = rp*(SP-up);

    //Compute acoustic star state
    ustar = (dp*up + dm*um + (-Ptotp + Ptotm))/(dp + dm);
    Ptotstar = (dp*Ptotm + dm*Ptotp + dp*dm*(um-up))/(dp + dm);

    //! Left star region variables
    rstarm = rm*(SM-um)/(SM-ustar);
    estar = rm*(SM-um)*(SM-ustar)-bm*bm;
    el = rm*(SM-um)*(SM-um)-bm*bm;

    if (abs(estar)<(1e-4)*bm*bm){
      Bt1starm = Bt1m;
      Bt2starm = Bt2m;
      vstarm = vm;
      wstarm = wm;
    }
    else{
      Bt1starm = Bt1m*el/estar;
      Bt2starm = Bt2m*el/estar;
      vstarm = vm - bm*Bt1m*(ustar - um)/estar;
      wstarm = wm - bm*Bt2m*(ustar - um)/estar;
    }
    estarm = ((SM-um)*Etotm - Ptotm*um + Ptotstar*ustar + bm*(um*bm + vm*Bt1m + wm*Bt2m - (ustar*bm + vstarm*Bt1starm + wstarm*Bt2starm)))/(SM-ustar);

    // Right star region variables
    rstarp = rp*(SP-up)/(SP-ustar);
    estar =  rp*(SP-up)*(SP-ustar)-bp*bp; //overwrite
    el = rp*(SP-up)*(SP-up)-bp*bp;

    if (abs(estar)<(1e-4)*bp*bp){
      Bt1starp = Bt1p;
      Bt2starp = Bt2p;
      vstarp = vp;
      wstarp = wp;
    }
    else{
      Bt1starp = Bt1p*el/estar;
      Bt2starp = Bt2p*el/estar;
      vstarp = vp - bp*Bt1p*(ustar - up)/estar;
      wstarp = wp - bp*Bt2p*(ustar - up)/estar;
    }
    estarp = ((SP-up)*Etotp - Ptotp*up + Ptotstar*ustar + bp*(up*bp + vp*Bt1p + wp*Bt2p - (ustar*bp + vstarp*Bt1starp + wstarp*Bt2starp)))/(SP-ustar);

    SAM = ustar - abs(bm)/sqrt(rstarm);
    SAP = ustar + abs(bp)/sqrt(rstarp);

    // double star
    vstarstar = (sqrt(rstarm)*vstarm + sqrt(rstarp)*vstarp + sg*(Bt1starp-Bt1starm))/(sqrt(rstarp)+sqrt(rstarm));
    wstarstar = (sqrt(rstarm)*wstarm + sqrt(rstarp)*wstarp + sg*(Bt2starp-Bt2starm))/(sqrt(rstarp)+sqrt(rstarm));
    Bt1starstar = (sqrt(rstarm)*Bt1starp + sqrt(rstarp)*Bt1starm + sg*sqrt(rstarp)*sqrt(rstarm)*(vstarp-vstarm))/(sqrt(rstarp)+sqrt(rstarm));
    Bt2starstar = (sqrt(rstarm)*Bt2starp + sqrt(rstarp)*Bt2starm + sg*sqrt(rstarp)*sqrt(rstarm)*(wstarp-wstarm))/(sqrt(rstarp)+sqrt(rstarm));

    estarstarm = estarm - sg*sqrt(rstarm)*((ustar*bm + vstarm*Bt1starm + wstarm*Bt2starm) - (ustar*bm + vstarstar*Bt1starstar + wstarstar*Bt2starstar));
    estarstarp = estarp + sg*sqrt(rstarp)*((ustar*bp + vstarp*Bt1starp + wstarp*Bt2starp) - (ustar*bp + vstarstar*Bt1starstar + wstarstar*Bt2starstar));

    //Sample the solution at x/t=0
      if(SM>0.0){
        wgdnv[0] = rm;
        wgdnv[dim1] = um;
        wgdnv[dim2] = vm;
        wgdnv[3] = wm;
        wgdnv[dim1 + 3] = bm;
        wgdnv[dim2 + 3] = Bt1m;
        wgdnv[6] = Bt2m;
        wgdnv[7] = Ptotm;
        etoto = Etotm;
        //vdotb = wgdnv[dim1]*wgdnv[dim1 + 3] + wgdnv[dim2]*wgdnv[dim2 + 3] + wgdnv[3]* wgdnv[6];
      }
      else if(SM<=0 && SAM>=0.0){
        wgdnv[0] = rstarm;
        wgdnv[dim1] = ustar;
        wgdnv[dim2] = vstarm;
        wgdnv[3] = wstarm;
        wgdnv[dim1 + 3] = bm;
        wgdnv[dim2 + 3] = Bt1starm;
        wgdnv[6] = Bt2starm;
        wgdnv[7] = Ptotstar;
        etoto = estarm;
        //vdotb = wgdnv[dim1]*wgdnv[dim1 + 3] + wgdnv[dim2]*wgdnv[dim2 + 3] + wgdnv[3]* wgdnv[6];
      }
      else if(ustar>=0.0 && SAM <= 0.0){
        wgdnv[0] = rstarm;
        wgdnv[dim1] = ustar;
        wgdnv[dim2] = vstarstar;
        wgdnv[3] = wstarstar;
        wgdnv[dim1 + 3] = bm;
        wgdnv[dim2 + 3] = Bt1starstar;
        wgdnv[6] = Bt2starstar;
        wgdnv[7] = Ptotstar;
        etoto = estarstarm;
        //vdotb = wgdnv[dim1]*wgdnv[dim1 + 3] + wgdnv[dim2]*wgdnv[dim2 + 3] + wgdnv[3]* wgdnv[6];
      }
      else if(SAP>=0.0 && ustar <= 0.0){
        wgdnv[0] = rstarm;
        wgdnv[dim1] = ustar;
        wgdnv[dim2] = vstarstar;
        wgdnv[3] = wstarstar;
        wgdnv[dim1 + 3] = bm;
        wgdnv[dim2 + 3] = Bt1starstar;
        wgdnv[6] = Bt2starstar;
        wgdnv[7] = Ptotstar;
        etoto = estarstarp;
        //vdotb = wgdnv[dim1]*wgdnv[dim1 + 3] + wgdnv[dim2]*wgdnv[dim2 + 3] + wgdnv[3]* wgdnv[6];
      }
      else if (SAP<=0.0 && SP >= 0.0){
        wgdnv[0] = rstarp;
        wgdnv[dim1] = ustar;
        wgdnv[dim2] = vstarp;
        wgdnv[3] = wstarp;
        wgdnv[dim1 + 3] = bp;
        wgdnv[dim2 + 3] = Bt1starp;
        wgdnv[6] = Bt2starp;
        wgdnv[7] = Ptotstar;
        etoto = estarp;
        //vdotb = wgdnv[dim1]*wgdnv[dim1 + 3] + wgdnv[dim2]*wgdnv[dim2 + 3] + wgdnv[3]* wgdnv[6];
      }
      else if (SP<=0.0){
        wgdnv[0] = rp;
        wgdnv[dim1] = up;
        wgdnv[dim2] = vp;
        wgdnv[3] = wp;
        wgdnv[dim1 + 3] = bp;
        wgdnv[dim2 + 3] = Bt1p;
        wgdnv[6] = Bt2p;
        wgdnv[7] = Ptotp;
        etoto = Etotp;
      }

    vdotb = wgdnv[dim1]*wgdnv[dim1 + 3] + wgdnv[dim2]*wgdnv[dim2 + 3] + wgdnv[3]* wgdnv[6];
    bnorm =  wgdnv[dim2 + 3]*wgdnv[dim2 + 3] + wgdnv[dim1 + 3]*wgdnv[dim1 + 3] + wgdnv[6]*wgdnv[6];

    FG[id]=wgdnv[0]*wgdnv[dim1];
    FG[id+size*dim1]=wgdnv[0]*wgdnv[dim1]*wgdnv[dim1]+wgdnv[7] - wgdnv[dim1 + 3]*wgdnv[dim1 + 3];
    FG[id+size*dim2]=wgdnv[0]*wgdnv[dim1]*wgdnv[dim2] - wgdnv[dim1 + 3]*wgdnv[dim2 + 3];
    FG[id+size*3]= wgdnv[0]*wgdnv[dim1]*wgdnv[3] - wgdnv[dim1 + 3]*wgdnv[6]; // z component
    FG[id+size*(dim1 + 3)]= 0.0; //!wgdnv[0]*wgdnv[dim1]; // B normal component
    FG[id+size*(dim2 + 3)]= wgdnv[dim1]*wgdnv[dim2+3] - wgdnv[dim2]*wgdnv[dim1+3];// wgdnv[0]*wgdnv[dim1]; // B tangential component
    FG[id+size*6]= wgdnv[dim1]*wgdnv[6] - wgdnv[3]*wgdnv[dim1+3]; // B normal component
    FG[id+size*7] = wgdnv[dim1]*(etoto+wgdnv[7]) - wgdnv[dim1+3]*(vdotb); //energy
  }
}

__global__ void flux_line_integral(double* edge,double* F,double* G,int m,int ny,int nx,int nvar){
  int id, ic, jc, im, jm, va;
  int q, idF, idG;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = nvar*d;
  double valx,valy;
  double chsi_m = -1, chsi_p = 1;
  valx=valy=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  va  = id/d;
  ic  = id - va*d;
  jm  = ic/c;
  ic -= jm*c;
  im  = ic/b;
  ic -= im*b;
  jc  = ic/a;
  ic -= jc*a;
  idF  = ic + jc*(nx+1) + va*m*ny*(nx+1);
  idG =  ic + jc*nx     + va*m*nx*(ny+1);
  b = (nx+1)*ny;
  c = (ny+1)*nx;
  if( id < size){
    for(q = 0; q < m; q++){
      valx += (F[q*b+idF+ 1]*legendre(chsi_p,im,1)-F[q*b+idF]*legendre(chsi_m,im,1))*legendre(yquad[q],jm,1)*wyquad[q];
      valy += (G[q*c+idG+nx]*legendre(chsi_p,jm,1)-G[q*c+idG]*legendre(chsi_m,jm,1))*legendre(xquad[q],im,1)*wxquad[q];
    }
    edge[id] = valx;
    edge[id+size] = valy;
  }
}


__global__ void flux_line_integral_ldf_t(double* edge,double* F,double* G, double invdx, double invdy, int m,int ny,int nx,int nvar){
  int id, ic, jc, im, jm, va;
  int q, idF, idG, idF2, idG2, mid;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int bsi = m*(m+3)/2;
  int modek;
  //int size = nvar*d;
  int size = bsi*nx*ny;
  double valx1,valy1,valx2,valy2;
  double chsi_m = -1, chsi_p = 1;
  valx1=valy1=valx2=valy2=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  modek  = id/(nx*ny);
  ic  = id - modek*nx*ny;
  jc  = ic/a;
  ic -= jc*a;

  idF  = ic + jc*(nx+1); // + va*m*ny*(nx+1);
  idG =  ic + jc*nx; //     + va*m*nx*(ny+1);

  idF2  = ic + jc*(nx+1) + m*ny*(nx+1);
  idG2 =  ic + jc*nx + m*nx*(ny+1);

  mid = modek;

  b = (nx+1)*ny;
  c = (ny+1)*nx;
  if( id < size){
    for(q = 0; q < m; q++){
      valx1 += (F[q*b+idF+ 1]*basis_ldf_t(chsi_p,yquad[q],mid,m,0)-F[q*b+idF]*basis_ldf_t(chsi_m,yquad[q],mid,m,0))*wyquad[q];
      valy1 += (G[q*c+idG+nx]*basis_ldf_t(xquad[q],chsi_p,mid,m,0)-G[q*c+idG]*basis_ldf_t(xquad[q],chsi_m,mid,m,0))*wxquad[q];

      valx2 += (F[q*b+idF2 + 1]*basis_ldf_t(chsi_p,yquad[q],mid,m,1)-F[q*b+idF2]*basis_ldf_t(chsi_m,yquad[q],mid,m,1))*wyquad[q];
      valy2 += (G[q*c+idG2+nx]*basis_ldf_t(xquad[q],chsi_p,mid,m,1)-G[q*c+idG2]*basis_ldf_t(xquad[q],chsi_m,mid,m,1))*wxquad[q];
    }
    edge[id] = invdx*(valx1 + valx2);
    edge[id + size] = invdy*(valy2 + valy1);
  }
}


__global__ void grad_phi(double* grad, double* x, double* y, double x0, double y0, double cutoff,  double eps, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  double dx, dy, r;
  if( id < size ){
    dx = x[id]-x0;
    dy = y[id]-y0;
    r = sqrt(dx*dx+dy*dy);
    cutoff=0.5-0.1*0.5;
    eps=0.25;
    if (r > cutoff){
      grad[id] = dx/(r*r*r);
      grad[id+size] = dy/(r*r*r);
      //grad[id] = dx/(r*(r*r+eps*eps));
      //grad[id+size] = dy/(r*(r*r+eps*eps));
    }
    else{
      //grad[id] = dx/(r*(r*r+eps*eps));
      //grad[id+size] = dy/(r*(r*r+eps*eps));
      grad[id] = dx/(r*(r*r+eps*eps))*(cutoff*(cutoff*cutoff+eps*eps))/(cutoff*cutoff*cutoff);
      grad[id+size] = dy/(r*(r*r+eps*eps))*(cutoff*(cutoff*cutoff+eps*eps))/(cutoff*cutoff*cutoff);
    }
  }
}

__global__ void grad_phi_soft(double* grad, double* x, double* y, double x0, double y0, double cutoff,  double eps, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  double dx, dy, r,vx,vy, vt;
  if( id < size ){
    dx = x[id]-x0;
    dy = y[id]-y0;
    r = sqrt(dx*dx+dy*dy);
    cutoff=0.25;//0.5-0.1*0.5;
    eps=0.1;
    grad[id] = dx/(r*(r*r+eps*eps)); //dx/(r*r*r);
    grad[id+size] = dy/(r*(r*r+eps*eps));
  }
}

#ifdef LASRC
__global__ void grad_phi_const(double* grad, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size )
       grad[id] = 1.0;
       grad[id+size] = 0.0;
}

#endif

__global__ void get_source(double* w, double* s, double* grad, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  double rho,gradx,grady;
  if( id < size ){
    rho = w[id];
    gradx = grad[id];
    grady = grad[id+size];
    s[id] = 0.0;
    s[id+size] = -rho*gradx;
    s[id+size*2] = -rho*grady;
    s[id+size*3] = -rho*(w[id+size]*gradx+w[id+size*2]*grady);
  }
}

__global__ void source_vol (double* s_vol, double* s, int m, int ny, int nx, int nvar) {
  int id, ic, jc, im, jm, va;
  int iq, jq, cid;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = nvar*d;
  double val=0.0;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  va  = id/d;
  ic  = id - va*d;
  jm  = ic/c;
  ic -= jm*c;
  im  = ic/b;
  ic -= im*b;
  jc  = ic/a;
  ic -= jc*a;
  cid = ic + jc*a + va*d;
  if( id < size ){
    for( iq=0; iq<m; iq++)
      for( jq=0; jq<m; jq++)
	val += s[iq*b+jq*c+cid]*legendre(xquad[iq],im,1)*wxquad[iq]*legendre(yquad[jq],jm,1)*wyquad[jq];
    s_vol[id] = val;
  }
}

__global__ void wave_killing_bc (double* nodes, double* x, double* y, double boxlen_x, double dt, int size) {
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  double lambdadt,rho0,xn,yn,r;
  if( id < size ){
    xn = x[id];
    yn = y[id];
    r = sqrt(xn*xn+yn*yn);
    rho0= 1./(1.+pow(r/(0.3*boxlen_x),10));
    //rho0=1.0;
    lambdadt = 0.005*dt*( r < 0.4*boxlen_x ?  0.0 : pow(1.-exp(-(r-0.4*boxlen_x)/(0.04*boxlen_x)),6.0));
    nodes[id] = (nodes[id]+lambdadt*rho0)/(1+lambdadt);
    //    nodes[id] = lambdadt;
  }
}

__global__ void HIO_limiter(double* modes, double* limited, int m, int ny, int nx, int nvar, int bc){
  int id,ic,jc,va,cid,lid,rid,bid,tid,i,j,done=0;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = nx*ny*nvar;
  double val1,val2,mode1,mode2;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  va  = id/b;
  ic  = id - va*b;
  jc  = ic/a;
  ic -= jc*a;
  cid =  ic + jc*a + va*d;
  if(id < size){
    lid = BC(ic-1,nx,bc) + jc*a + va*d;
    rid = BC(ic+1,nx,bc) + jc*a + va*d;
    tid = ic + BC(jc+1,ny,bc)*a + va*d;
    bid = ic + BC(jc-1,ny,bc)*a + va*d;
    for(i=m-1;i>0;i--){
      val1 = limiting(modes,cid,lid,rid,bid,tid,i,i,m,b);
      limited[(i+i*m)*b+cid] = val1;//modes[(i+i*m)*b+cid];
      mode1 = modes[(i+i*m)*b+cid];
      if (fabs(val1 - mode1) < PRC*fabs(mode1))
      	break;
      //limited[(i+i*m)*b+cid] = val1;
      for(j=i-1;j>=0;j--){
      	val1 = limiting(modes,cid,lid,rid,bid,tid,i,j,m,b);
      	val2 = limiting(modes,cid,lid,rid,bid,tid,j,i,m,b);
      	limited[(i+j*m)*b+cid] = val1;//modes[(i+j*m)*b+cid];
      	limited[(j+i*m)*b+cid] = val2;//modes[(j+i*m)*b+cid];
      	//mode1 = modes[(i+j*m)*b+cid];
      	//mode2 = modes[(j+i*m)*b+cid];
      	if (fabs(val1 - mode1) < PRC*fabs(mode1) && fabs(val2 - mode2) < PRC*fabs(mode2)){
      	  done = 1;
      	  break;
      	}
    	//limited[(i+j*m)*b+cid] = val1;
    	//limited[(j+i*m)*b+cid] = val2;
      }
      if(done == 1)
	       break;
    }
  }
}

__global__ void limit_rho(double* modes, double* uX, double* uY, double* pmodes, double eps, int k, int m, int ny, int nx, int nvar){
  int id, iq, jq, gll, gl;
  int a = nx;
  int b = ny*a;
  int size = nx*ny;
  double theta,rho_av,rho_min,valx,valy;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size ){
    rho_av = modes[id];
    rho_min = rho_av;
   for(gll=0;gll<k;gll++){
      for(gl=0;gl<m;gl++){
	valx=uX[(gl+gll*m)*b+id];
	valy=uY[(gl+gll*m)*b+id];
	if(valx < rho_min)
	  rho_min = valx;
	if(valy < rho_min)
	  rho_min = valy;
      }
    }
    theta = min(fabs((rho_av-eps)/(rho_av-rho_min)),1.);
    if(theta<1.)
      for(iq=0;iq<m;iq++)
	for(jq=0;jq<m;jq++)
	  if (iq+jq>0)
	    pmodes[(iq+jq*m)*b+id] = theta*modes[(iq+jq*m)*b+id];
  }
}

__global__ void limit_by_pressure(double* uX, double* uY, double* pmodes,  double* modes, double gamma, double eps, int k, int m, int ny, int nx, int nvar){
  int id, va, gl, gll;
  int im, jm, qid, xs;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;

  int size = nx*ny;
  double tau, tau_min=1.0, P, rho, Mx, My, e, rhoav, mxav, myav, eav;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  xs = m*k*ny*nx;
  if( id < size ){
    rhoav = pmodes[id];
    mxav  = pmodes[id+d];
    myav  = pmodes[id+d*2];
    eav   = pmodes[id+d*3];

    for(gll=0;gll<k;gll++){
      for(gl=0;gl<m;gl++){
	qid = (gl+gll*m)*b+id;
	rho = uX[qid];
	Mx  = uX[qid+xs];
	My  = uX[qid+xs*2];
	e   = uX[qid+xs*3];
	P = (gamma-1.)*(e-0.5*(Mx*Mx+My*My)/rho);
	if(P >= eps)
	  tau = 1.;
	else
	  tau = solve_for_t(rho,Mx,My,e,rhoav,mxav,myav,eav,gamma,eps,id);
	if(tau < tau_min)
	  tau_min = tau;
	rho = uY[qid];
	Mx  = uY[qid+xs];
	My  = uY[qid+xs*2];
	e   = uY[qid+xs*3];
	P = (gamma-1.)*(e-0.5*(Mx*Mx+My*My)/rho);
	if(P >= eps)
	  tau = 1.;
	else
	  tau = solve_for_t(rho,Mx,My,e,rhoav,mxav,myav,eav,gamma,eps,id);
	if(tau < tau_min)
	  tau_min = tau;
      }
    }
    for(va = 0; va < nvar; va++)
      for( im=0; im < m; im++)
	for( jm=0; jm < m; jm++)
	  if (im+jm>0)
	    modes[(im+jm*m)*b+id+va*d] = tau_min*pmodes[(im+jm*m)*b+id+va*d];
  }
}
__global__ void check_positivity(double* uX, double* uY, double* modes, double gamma, double eps, int k, int m, int ny, int nx, int nvar){
  int id, va, gl, gll;
  int im, jm, qid, xs, negative;
  int a = nx;
  int b = ny*a;
  int c = m*b;
  int d = m*c;
  int size = nx*ny;
  double P, rho, vx, vy;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  xs = m*k*ny*nx;
  if( id < size ){
    for(gll=0;gll<k;gll++){
      for(gl=0;gl<m;gl++){
	qid = (gl+gll*m)*b+id;
	rho = uX[qid];
	vx  = uX[qid+xs]/rho;
	vy  = uX[qid+xs*2]/rho;
	P = (gamma-1.)*(uX[qid+xs*3]-0.5*(vx*vx+vy*vy)*rho);
	if( P < eps || rho < eps ){
	   negative = 1;
	   break;
	}
	rho = uY[qid];
	vx  = uY[qid+xs]/rho;
	vy  = uY[qid+xs*2]/rho;
	P = (gamma-1.)*(uY[qid+xs*3]-0.5*(vx*vx+vy*vy)*rho);
	if( P < eps || rho < eps ){
	   negative = 1;
	   break;
	}
      }
    }
    if( negative == 1)
      for(va = 0; va < nvar; va++)
	for( im=0; im < m; im++)
	  for( jm=0; jm < m; jm++)
	    if (im+jm>0)
	      modes[(im+jm*m)*b+id+va*d] = 0.0;
  }
}

__global__ void compute_dudt_ldf_t(double* dudt, double* flux_vol, double* edges, double invdx, double invdy, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size ){
    dudt[id] = 0.5*(flux_vol[id] - edges[id] - edges[id+size]);
  }

}

__global__ void compute_dudt(double* dudt, double* flux_vol, double* edges, double* src_vol, double invdx, double invdy, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size ){
    dudt[id] = 0.5*(flux_vol[id] - invdx*edges[id] - invdy*edges[id+size]);
  }

}

__global__ void compute_dudt_b(double* dudt, double* flux_vol, double* edges, double invdx, double invdy, int size){
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size ){
    dudt[id] = 0.5*(flux_vol[id]- invdx*edges[id] - invdy*edges[id+size]);
  }

}

__global__ void sum3 (double* out, double* A, double* B, double* C, double alpha, double beta, double gamma, int size)
{
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size )
    out[id] = A[id]*alpha + B[id]*beta + C[id]*gamma;
}

__global__ void minus2 (double* out, double* A, double* B, int size)
{
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size )
    out[id] = A[id] - B[id];
}

__global__ void plus_equal (double* out, double* A, double* B, double alpha, double beta, int size)
{
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size )
    out[id] += A[id]*alpha + B[id]*beta;
}
__global__ void sum2 (double* out, double* A, double* B, double beta, int size)
{
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size )
    out[id] = A[id] + B[id]*beta;
}

__global__ void sumequal (double* out, double* A, double* B, int size)
{
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size )
    out[id] = A[id] + B[id];
}

__global__ void timesC (double* out, double* A, double constant, int size)
{
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size )
    out[id] = A[id]*constant;
}

__global__ void parabolic_decay (double* out, double* A, double constant, int size)
{
  int id;
  id = blockDim.x * blockIdx.x + threadIdx.x;
  if( id < size )
    out[id] = A[id]*constant;
}

extern "C" void device_get_modes_from_nodes_(double** nodes, double** modes){
  int size = nx*ny*m*m*nvar;
  get_modes_from_nodes<<<(size+BLOCK-1)/BLOCK,BLOCK>>>(*nodes,*modes,m,ny,nx,nvar);
}


extern "C" void device_get_nodes_from_modes_(double** modes, double** nodes){
  int size = nx*ny*m*m*nvar;
  get_nodes_from_modes<<<(size+BLOCK-1)/BLOCK,BLOCK>>>(*modes,*nodes,m,ny,nx,nvar);
}

extern "C" void device_get_modes_from_nodes_ldf_b_2_(double** nodes, double** modes, double** bmodes){
  int bsize = m*(m+3)/2;
  get_modes_from_nodes_ldf_test_2<<<(nx*ny*bsize+BLOCK-1)/BLOCK,BLOCK>>>(*nodes,*bmodes,m,ny,nx,nvar);

}

extern "C" void device_get_nodes_from_modes_ldf_b_2_(double** modes, double** bmodes, double** nodes){
  int usize = nx*ny*m*m;
  get_nodes_from_modes_ldf_test_2<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(*bmodes,*nodes,m,ny,nx,nvar);
}

extern "C" void device_compute_min_dt_t_ (double* Dt){
  int size = nx*ny;
  int usize = nx*ny*m*m;
  double dt;
  // get nodes from modes
  get_nodes_from_modes_ldf_test_2<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(b_modes,w1,m,ny,nx,nvar);
  compute_primitive_t<<<(size+BLOCK-1)/BLOCK,BLOCK>>>( w1, w, gmma, m,usize, size);
  compute_min_dt<<<1,BLOCK>>>( w, pivot, gmma, cfl, dx, dy, m, usize, size);
  hipMemcpy(&dt,pivot,sizeof(double),hipMemcpyDeviceToHost);
  *Dt = dt;
}

extern "C" void device_compute_min_dt_ (double* Dt){
  int size = nx*ny;
  double dt;
  compute_primitive<<<(size+BLOCK-1)/BLOCK,BLOCK>>>( du, w, gmma, usize, size);
  compute_min_dt<<<1,BLOCK>>>( w, pivot, gmma, cfl, dx, dy, m, usize, size);
  hipMemcpy(&dt,pivot,sizeof(double),hipMemcpyDeviceToHost);
  *Dt = dt;
}

__global__ void compute_max_v(double* w, double* vm, double gamma, double cfl, double dx, double dy, int m, int usize, int size){
  int id, jump;
  double dt,dt_min,csx,csy,constant,cs,d2, v, vmax;
  __shared__ double maxs[BLOCK];
  id = threadIdx.x;


  if(id < size){
    cs = gamma;
    d2 = 0.5*((w[id]*w[id] + w[id+usize]*w[id+usize]+w[id+usize*2]*w[id+usize*2])/1.0 + cs);
    csx = sqrt(d2 + sqrt(d2*d2-cs*1.0));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
    csy = sqrt(d2 + sqrt(d2*d2-cs*1.0));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));

    vmax = 1.0; //sqrt(pow((fabs(1.0)),2.)+pow((fabs(1.0)),2.));
    for (id = threadIdx.x+blockDim.x; id < size; id += blockDim.x){ //This is implemented considering only one block in the reduction launch.
      cs = gamma;
      d2 = 0.5*((w[id]*w[id] + w[id+usize]*w[id+usize]+w[id+usize*2]*w[id+usize*2])/1.0 + cs);
      csx = sqrt(d2 + sqrt(d2*d2-cs*1.0));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
      csy = sqrt(d2 + sqrt(d2*d2-cs*1.0));//sqrt(gamma*max(w[id+usize*3],P0)/max(w[id],RHO0));
      v = 1.0; //sqrt(pow((fabs(1.0)),2.)+pow((fabs(1.0)),2.));
      vmax=max(v,vmax);
    }
    maxs[threadIdx.x] = vmax;
  }
  __syncthreads();
  for(jump = blockDim.x/2; jump > 0; jump >>= 1){
    if( threadIdx.x < jump )
      maxs[threadIdx.x]=max(maxs[threadIdx.x],maxs[threadIdx.x+jump]);
    __syncthreads();
  }
  if(threadIdx.x == 0)
    vm[0] = maxs[0];
}

extern "C" void device_compute_max_v_(double *vm){
  int size = nx*ny;
  double v;
  compute_primitive<<<(size+BLOCK-1)/BLOCK,BLOCK>>>( du, w, gmma, usize, size);
  compute_max_v<<<1,BLOCK>>>( w, pivot, gmma, cfl, dx, dy, m, usize, size);
  hipMemcpy(&v,pivot,sizeof(double),hipMemcpyDeviceToHost);
  *vm = v;

}

extern "C" void device_compute_limiter_(double** modes){
#ifdef LIMIT
    int size=nx*ny*nvar;
  #ifdef TVD
    compute_primitive<<<(nx*ny+BLOCK-1)/BLOCK,BLOCK>>>(*modes,w,gmma,usize,nx*ny);
    size = nx*ny*(m*m-1);
    cons_to_prim<<<(size+BLOCK-1)/BLOCK,BLOCK>>>(*modes,w,gmma,m,ny,nx,usize,size);
    hipMemcpy(pivot,w,tsize*sizeof(double),hipMemcpyDeviceToDevice);
    HIO_limiter<<<(nx*ny*nvar+BLOCK-1)/BLOCK,BLOCK>>>(pivot,w,m,ny,nx,nvar,bc);
    prim_to_cons<<<(size+BLOCK-1)/BLOCK,BLOCK>>>(w,*modes,gmma,m,ny,nx,usize,size);
  #endif

  #ifdef HIO
    compute_primitive<<<(nx*ny+BLOCK-1)/BLOCK,BLOCK>>>(*modes,w,gmma,usize,nx*ny);
    size = nx*ny*(m*m-1);
    cons_to_prim<<<(size+BLOCK-1)/BLOCK,BLOCK>>>(*modes,w,gmma,m,ny,nx,usize,size);
    hipMemcpy(pivot,w,tsize*sizeof(double),hipMemcpyDeviceToDevice);
    HIO_limiter<<<(nx*ny*nvar+BLOCK-1)/BLOCK,BLOCK>>>(pivot,w,m,ny,nx,nvar,bc);
    prim_to_cons<<<(size+BLOCK-1)/BLOCK,BLOCK>>>(w,*modes,gmma,m,ny,nx,usize,size);
  #endif
  #ifdef CP
    double eps = 1E-10;
    compute_GxGLL<<<(nx*ny*m*k+BLOCK-1)/BLOCK,BLOCK>>>(uX,uY,*modes,k,m,ny,nx,nvar);
    check_positivity<<<(nx*ny+BLOCK-1)/BLOCK,BLOCK>>>(uX,uY,*modes,gmma,eps,k,m,ny,nx,nvar);
  #endif
  #ifdef PP
    compute_GxGLL<<<(nx*ny*m*k+BLOCK-1)/BLOCK,BLOCK>>>(uX,uY,*modes,k,m,ny,nx,1);//Only for the density
    hipMemcpy(pivot,*modes,tsize*sizeof(double),hipMemcpyDeviceToDevice);
    limit_rho<<<(nx*ny+BLOCK-1)/BLOCK,BLOCK>>>(*modes,uX,uY,pivot,RHO0,k,m,ny,nx,nvar);
    compute_GxGLL<<<(nx*ny*m*k*nvar+BLOCK-1)/BLOCK,BLOCK>>>(uX,uY,pivot,k,m,ny,nx,nvar);//Now for all the quantities with the positive modes of the density
    limit_by_pressure<<<(nx*ny+BLOCK-1)/BLOCK,BLOCK>>>(uX,uY,pivot,*modes,gmma,P0,k,m,ny,nx,nvar);
    hipDeviceSynchronize();
  #endif
#endif
}

extern "C" void compute_psi_correction(double* dudt, double* modes, double dt, double vm){
  double ch, cp2;
  //vm = 0.5;
  //printf("velo: %.14f \n", vm);
  ch = vm*5.;//cbrt(0.2*1.0/(dt*dt));
  cp2 = ch*0.18;
  // convert psi to nodes (change to only do the last variable)
  get_nodes_from_modes<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(modes,u_d_q,m,ny,nx,nvar);
  // perform integral \int psi * phi dxdy
  //volume_integral<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(pivot,u_d_q,-(ch*ch)/(cp2),invdx,invdy,m,ny,nx,nvar);
  // add correction to
  volume_integral<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(pivot,u_d_q,-1.0/(cp2),invdx,invdy,m,ny,nx,nvar);

  sum2<<< (usize+BLOCK-1)/BLOCK,BLOCK >>>(&dudt[usize*3],&dudt[usize*3], &pivot[usize*3], ch*ch, usize);

  hipDeviceSynchronize();
}

extern "C" void device_compute_update_(int* Iter, int* SSP, double* DT, double* T, double *vm){
  double dt = *DT;
  double t = *T;
  int iter = *Iter;
  int RK = *SSP;
  double v_max = *vm;
  double* modes;
  double ch;
#if defined(SRC) && defined(PLANET)
  double delta_r = 0.1;
  double cutoff = 0.5-0.5*delta_r;
  double eps = 0.1;
#endif

  switch (iter){
  case 0:
    modes = du;
    break;
  case 1:
    modes = w1;
    break;
  case 2:
    modes = w2;
    break;
  case 3:
    modes = w3;
    break;
  case 4:
    modes = w4;
    break;
  }
  //device_compute_limiter_(&modes);

  // slve parabolic part of psi


  get_nodes_from_modes<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(modes,u_d_q,m,ny,nx,nvar);
  compute_primitive<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>( u_d_q, w, gmma, usize, usize);
  compute_flux<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(u_d_q, w, flux_q1, flux_q2, usize);

  flux_vol<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(flux_v,flux_q1,flux_q2,invdx,invdy,m,ny,nx,nvar);
  compute_faces<<<(nx*ny*nvar+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,modes,m,ny,nx,nvar);
  compute_primitive<<<(4*nx*ny*m+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,gmma,4*nx*ny*m,4*nx*ny*m);

#ifdef UPWIND
  compute_flux<<<(4*nx*ny*m+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,flux_f2,4*nx*ny*m);
  compute_true_upwind<<<(m*ny*(nx+1)+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,F,gmma,m,ny,nx,nvar,0,bc,m*ny*(nx+1));
  compute_true_upwind<<<(m*(ny+1)*nx+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f2,G,gmma,m,ny,nx,nvar,1,bc,m*(ny+1)*nx);
#else
  #ifndef HLLD
    compute_flux<<<(4*nx*ny*m+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,flux_f2,4*nx*ny*m);
    compute_LLF<<<(m*ny*(nx+1)+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,F,gmma,m,ny,nx,nvar,0,bc,m*ny*(nx+1));
    compute_LLF<<<(m*(ny+1)*nx+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f2,G,gmma,m,ny,nx,nvar,1,bc,m*(ny+1)*nx);
  #else
    compute_HLLD<<<(m*ny*(nx+1)+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,F,gmma,m,ny,nx,nvar,0,bc,m*ny*(nx+1));
    compute_HLLD<<<(m*(ny+1)*nx+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,G,gmma,m,ny,nx,nvar,1,bc,m*(ny+1)*nx);
  #endif
#endif

  flux_line_integral<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(edge,F,G,m,ny,nx,nvar);

  compute_dudt<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(dudt,flux_v,edge,src_vol,invdx,invdy,nx*ny*m*m*nvar);

  // add hyperbolic cleaning
  //#ifdef CORR2
   //v_max = device_compute_max_v()*7.0;
   //ch = v_max*5.;
   //timesC<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(&dudt[usize*3],&dudt[usize*3],ch*ch,usize);
   //hipDeviceSynchronize();
   //compute_psi_correction(dudt,modes,dt,v_max);
   //hipDeviceSynchronize();

   //printf("ch: %.14f \n", ch);
   //timesC<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(&dudt[usize*3],&dudt[usize*3],ch*ch,usize);
   //hipDeviceSynchronize();
   //compute_psi_correction(dudt,modes,dt,v_max);
   //hipDeviceSynchronize();
  //#endif

  if (RK==4){
    switch (iter){
    case 0:
      sum2<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w1,du,dudt,(double)0.391752226571890*dt, tsize);
      break;
    case 1:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w2,du,w1,dudt,0.444370493651235,0.555629506348765,0.368410593050371*dt, tsize);
      break;
    case 2:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w3,du,w2,dudt,0.620101851488403,0.379898148511597,0.251891774271694*dt, tsize);
      break;
    case 3:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w4,du,w3,dudt,0.178079954393132,0.821920045606868,0.544974750228521*dt, tsize);
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,w2,w3,dudt,0.517231671970585,0.096059710526147,0.063692468666290*dt, tsize);
      break;
    case 4:
      plus_equal<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,w4,dudt,0.386708617503269,0.226007483236906*dt, tsize);
      hipDeviceSynchronize();
      break;
    }
  }
  else if(RK==3){
    switch (iter){
    case 0:
      sum2<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w1,du,dudt,dt,tsize);
      break;
    case 1:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w2,du,w1,dudt,0.75,0.25,0.25*dt, tsize);
      break;
    case 2:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,du,w2,dudt,1.0/3.0,2.0/3.0,2.0/3.0*dt, tsize);
      hipDeviceSynchronize();
      break;
     }
  }
  else if(RK==2){
    switch (iter){
    case 0:
      sum2<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w1,du,dudt,dt,tsize);
      break;
    case 1:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,du,w1,dudt,0.5,0.5,0.5*dt, tsize);
      hipDeviceSynchronize();
      break;
    }
  }
  else if(RK==1){
    switch (iter){
    case 0:
      sum2<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,du,dudt,dt,tsize);
      hipDeviceSynchronize();
      break;
    }
  }
}

extern "C" void device_compute_update_lldf_test_new_(int* Iter, int* SSP, double* DT, double* T, double *vm){
  double dt = *DT;
  double t = *T;
  int iter = *Iter;
  int RK = *SSP;
  int bs = nx*ny*m*(m+3)/2;
  double v_max = *vm;
  double* modes;
  double* bmodes;

  switch (iter){
  case 0:
    modes = du;
    bmodes = b_modes;
    break;
  case 1:
    modes = w1;
    bmodes = b_modes1;
    break;
  case 2:
    modes = w2;
    bmodes= b_modes2;
    break;
  case 3:
    modes = w3;
    bmodes= b_modes3;
    break;
  case 4:
    modes = w4;
    bmodes= b_modes4;
    break;
  }
  //device_compute_limiter_(&modes);

  get_nodes_from_modes_ldf_test_2<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(bmodes,u_d_q,m,ny,nx,nvar); // TODO
  compute_primitive<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>( u_d_q, w, gmma, usize, usize);

  compute_flux<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(u_d_q, w, flux_q1, flux_q2, usize);

  flux_vol_ldf_t<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(flux_v_b,flux_q1,flux_q2,invdx,invdy,m,ny,nx,nvar); // TODO

  compute_faces_ldf_t<<<(nx*ny+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,bmodes,m,ny,nx,nvar); // TODO

  compute_primitive<<<(4*nx*ny*m+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,gmma,4*nx*ny*m,4*nx*ny*m);

#ifdef UPWIND
  compute_flux<<<(4*nx*ny*m+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,flux_f2,4*nx*ny*m);
  compute_upwind<<<(m*ny*(nx+1)+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,F,gmma,m,ny,nx,nvar,0,bc,m*ny*(nx+1));
  compute_upwind<<<(m*(ny+1)*nx+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f2,G,gmma,m,ny,nx,nvar,1,bc,m*(ny+1)*nx);
#else
  #ifndef HLLD
    compute_flux<<<(4*nx*ny*m+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,flux_f2,4*nx*ny*m);
    compute_LLF<<<(m*ny*(nx+1)+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,F,gmma,m,ny,nx,nvar,0,bc,m*ny*(nx+1));
    compute_LLF<<<(m*(ny+1)*nx+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f2,G,gmma,m,ny,nx,nvar,1,bc,m*(ny+1)*nx);
  #else
    compute_HLLD<<<(m*ny*(nx+1)+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,F,gmma,m,ny,nx,nvar,0,bc,m*ny*(nx+1));
    compute_HLLD<<<(m*(ny+1)*nx+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,G,gmma,m,ny,nx,nvar,1,bc,m*(ny+1)*nx);
  #endif
#endif

  flux_line_integral_ldf_t<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(edges_b,F,G,invdx,invdy,m,ny,nx,nvar); // TODO

  compute_dudt_ldf_t<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(dudt_b,flux_v_b,edges_b,invdx,invdy,bs); // TODO

  hipDeviceSynchronize();

  if(RK==2){
    switch (iter){
    case 0:
      sum2<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes1,b_modes,dudt_b,dt,bs); // TODO
      break;
    case 1:
      sum3<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes,b_modes,b_modes1,dudt_b,0.5,0.5,0.5*dt, bs); // TODO
      hipDeviceSynchronize();
      break;
    }
  }
  else if(RK==3){
    switch (iter){
    case 0:
      sum2<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes1,b_modes,dudt_b,dt,bs);
      break;
    case 1:
      sum3<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes2,b_modes,b_modes1,dudt_b,0.75,0.25,0.25*dt, bs);
      break;
    case 2:
      sum3<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes,b_modes,b_modes2,dudt_b,1.0/3.0,2.0/3.0,2.0/3.0*dt, bs);
      hipDeviceSynchronize();
      break;
     }
  }
  else if(RK==1){
    switch (iter){
    case 0:
      sum2<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes,b_modes,dudt_b,dt,bs); // TODO
      hipDeviceSynchronize();
      break;
    }
  }
  else if(RK==4){
    switch (iter){
    case 0:
      sum2<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes1,b_modes,dudt_b,(double)0.391752226571890*dt, bs);
      break;
    case 1:
      sum3<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes2,b_modes,b_modes1,dudt_b,0.444370493651235,0.555629506348765,0.368410593050371*dt, bs);
      break;
    case 2:
      sum3<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes3,b_modes,b_modes2,dudt_b,0.620101851488403,0.379898148511597,0.251891774271694*dt, bs);
      break;
    case 3:
      sum3<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes4,b_modes,b_modes3,dudt_b,0.178079954393132,0.821920045606868,0.544974750228521*dt, bs);
      sum3<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes,b_modes2,b_modes3,dudt_b,0.517231671970585,0.096059710526147,0.063692468666290*dt, bs);
      break;
    case 4:
      plus_equal<<<(bs+BLOCK-1)/BLOCK,BLOCK>>>(b_modes,b_modes4,dudt_b,0.386708617503269,0.226007483236906*dt, bs);
      hipDeviceSynchronize();
      break;
    }
  }

}

extern "C" void post_process_b_(int* Iter, int* SSP, double* DT, double* T, double *vm, double *CH){
  double dt = *DT;
  double t = *T;
  int iter = *Iter;
  int RK = *SSP;
  double v_max = *vm;
  double ch = *CH;
  double dx = 1./nx;
  double* modes;

  switch (iter){
  case 0:
    modes = du;
    break;
  case 1:
    modes = w1;
    break;
  case 2:
    modes = w2;
    break;
  case 3:
    modes = w3;
    break;
  case 4:
    modes = w4;
    break;
  }

  get_nodes_from_modes<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(modes,u_d_q,m,ny,nx,nvar);
  compute_primitive<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>( u_d_q, w, gmma, usize, usize);
  compute_flux_b<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(u_d_q, w, flux_q1, flux_q2, ch, m, usize);

  flux_vol<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(flux_v,flux_q1,flux_q2,invdx,invdy,m,ny,nx,nvar);
  compute_faces<<<(nx*ny*nvar+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,modes,m,ny,nx,nvar);
  compute_primitive<<<(4*nx*ny*m+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,gmma,4*nx*ny*m,4*nx*ny*m);

  compute_flux_b<<<(4*nx*ny*m+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,flux_f2, ch, m, 4*nx*ny*m);
  compute_upwind<<<(m*ny*(nx+1)+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,F,gmma,m,ny,nx,nvar,0,bc,m*ny*(nx+1));
  compute_upwind<<<(m*(ny+1)*nx+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f2,G,gmma,m,ny,nx,nvar,1,bc,m*(ny+1)*nx);
  //compute_average<<<(m*ny*(nx+1)+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f1,F,gmma,m,ny,nx,nvar,0,bc,m*ny*(nx+1));
  //compute_average<<<(m*(ny+1)*nx+BLOCK-1)/BLOCK,BLOCK>>>(ufaces,wfaces,flux_f2,G,gmma,m,ny,nx,nvar,1,bc,m*(ny+1)*nx);

  flux_line_integral<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(edge,F,G,m,ny,nx,nvar);

  compute_dudt_b<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(dudt,flux_v,edge,invdx,invdy,nx*ny*m*m*nvar);

  if (RK==4){
    switch (iter){
    case 0:
      sum2<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w1,du,dudt,(double)0.391752226571890*dt, tsize);
      break;
    case 1:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w2,du,w1,dudt,0.444370493651235,0.555629506348765,0.368410593050371*dt, tsize);
      break;
    case 2:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w3,du,w2,dudt,0.620101851488403,0.379898148511597,0.251891774271694*dt, tsize);
      break;
    case 3:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w4,du,w3,dudt,0.178079954393132,0.821920045606868,0.544974750228521*dt, tsize);
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,w2,w3,dudt,0.517231671970585,0.096059710526147,0.063692468666290*dt, tsize);
      break;
    case 4:
      plus_equal<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,w4,dudt,0.386708617503269,0.226007483236906*dt, tsize);
      hipDeviceSynchronize();
      break;
    }
  }
  else if(RK==3){
    switch (iter){
    case 0:
      sum2<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w1,du,dudt,dt,tsize);
      break;
    case 1:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w2,du,w1,dudt,0.75,0.25,0.25*dt, tsize);
      break;
    case 2:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,du,w2,dudt,1.0/3.0,2.0/3.0,2.0/3.0*dt, tsize);
      hipDeviceSynchronize();
      break;
     }
  }
  else if(RK==2){
    switch (iter){
    case 0:
      sum2<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(w1,du,dudt,dt,tsize);
      break;
    case 1:
      sum3<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,du,w1,dudt,0.5,0.5,0.5*dt, tsize);
      hipDeviceSynchronize();
      break;
    }
  }
  else if(RK==1){
    switch (iter){
    case 0:
      sum2<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(du,du,dudt,dt,tsize);
      hipDeviceSynchronize();
      break;
    }
  }
}

extern "C" void parabolic_psi_(double* DT, double* T, double *vm, double *Dx, double *Coeff){
  double dt = *DT;
  double t = *T;
  double dx = *Dx;
  double v_max = *vm;
  double* modes;
  double constant;
  double coeff = *Coeff;
  modes = du;

  //parabolic_term<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(du,du,v_max,dt);
  //ch = 0.15/(2.0*m-1)*(dx/(2.0*dt)); // v_max;
  //printf("%f %f\n",ch, cp );
  //cp2 = dx*ch/0.4;
  //constant = exp(-3.*dt);
  //constant = exp(-0.2*ch/(dx)*dt);
  //coeff = 0.4*ch/(dx/dt);
  constant = exp(-coeff);

  //constant = exp(-(ch*ch/cp2)*dt);

  //get_nodes_from_modes<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(modes,u_d_q,m,ny,nx,nvar);

  //hipDeviceSynchronize();
  parabolic_decay<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(&modes[usize*3],&modes[usize*3],constant,usize);

  hipDeviceSynchronize();
  //get_modes_from_nodes<<<(tsize+BLOCK-1)/BLOCK,BLOCK>>>(u_d_q,modes,m,ny,nx,nvar);

  //hipDeviceSynchronize();
}

extern "C" void mem_check(size_t free, size_t total){
  //size_t free;
  //size_t total;
  hipError_t error = hipMemGetInfo(&free,&total);
  hipError_t error2 = hipGetLastError();
  printf("free: %ld, total: %ld \n",free, total);
  if(error2 != hipSuccess){
    printf("Mem_check_error: %s\n", hipGetErrorString(error2));
  }

}

template < typename T >
inline void __checkCudaErrors(T result, char const *const func, const char *const file, int const line)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
    {
        fprintf(stderr, "%s:%i : checkCudaErrors() CUDA error (#%d): %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(-1);
    }
}

#define CCE(val) __checkCudaErrors( (val), #val, __FILE__, __LINE__ )


extern "C" void gpu_allocation_ (int *Nvar, int* Nx, int* Ny, int* M, int* K, double *Bl_x, double *Bl_y, double *CFL, double *Eta, int* Bc, int* nequilibrium, double *Gamma) {
  size_t free;
  size_t total;
  nvar = *Nvar;
  nx = *Nx;
  ny = *Ny;
  m = *M;
  k = *K;
  bc = *Bc;
  neql = *nequilibrium;
  usize = m*m*nx*ny;
  tsize = usize*nvar;
  boxlen_x = *Bl_x;
  boxlen_y = *Bl_y;
  dx = boxlen_x/double(nx);
  dy = boxlen_y/double(ny);
  invdx = 1/dx;
  invdy = 1/dy;
  cfl = *CFL;
  gmma = *Gamma;
  eta = *Eta;
  vm = 0;
  bsize = m*(m+3)/2;
  //hipError_t error = hipGetLastError();
  //hipError_t memerror = hipMemGetInfo(&free,&total);
  //error = hipGetLastError();
  hipMalloc ( &u, tsize * sizeof(double));
  hipMalloc ( &u_eq, tsize * sizeof(double));
  hipMalloc ( &u_d_q, tsize * sizeof(double));
  hipMalloc ( &du, tsize * sizeof(double));
  hipMalloc ( &w, tsize * sizeof(double));
  hipMalloc ( &w1, tsize * sizeof(double));
  hipMalloc ( &w2, tsize * sizeof(double));
  hipMalloc ( &w3, tsize * sizeof(double));
  hipMalloc ( &w4, tsize * sizeof(double));
  hipMalloc ( &dudt, tsize * sizeof(double));
  hipMalloc ( &ufaces_eq, 4*nvar*nx*ny*m*sizeof(double));

  hipMalloc ( &b_modes, bsize*nx*ny * sizeof(double));
  hipMalloc ( &b_modes1, bsize*m*nx*ny * sizeof(double));
  hipMalloc ( &b_modes2, bsize*m*nx*ny * sizeof(double));
  hipMalloc ( &b_modes3, bsize*m*nx*ny * sizeof(double));
  hipMalloc ( &b_modes4, bsize*m*nx*ny * sizeof(double));

  hipMalloc ( &flux_v_b,  bsize*nx*ny * sizeof(double));
  hipMalloc ( &edges_b, 2*bsize*nx*ny * sizeof(double));
  hipMalloc ( &dudt_b, bsize*nx*ny * sizeof(double));

  hipMalloc ( &ufaces, 4*nvar*nx*ny*m*sizeof(double));
  hipMalloc ( &wfaces, 4*nvar*nx*ny*m*sizeof(double));
  hipMalloc ( &flux_f1, 4*nvar*nx*ny*m*sizeof(double));
  hipMalloc ( &flux_f2, 4*nvar*nx*ny*m*sizeof(double));
  hipMalloc ( &flux_q1, tsize * sizeof(double));
  hipMalloc ( &flux_q2, tsize * sizeof(double));
  hipMalloc ( &flux_v,  tsize * sizeof(double));
  hipMalloc ( &F, nvar*(nx+1)*ny*m * sizeof(double));
  hipMalloc ( &G, nvar*nx*(ny+1)*m * sizeof(double));
  hipMalloc ( &edge, tsize*2 * sizeof(double));
  /*error = hipGetLastError();
  if(error != hipSuccess){
    printf("Error1");
    printf("CUDA error gpu init: %s\n", hipGetErrorString(error));
    exit(-1);
  }*/



#if defined(SRC) || defined(LASRC)
  hipMalloc ( &src, tsize * sizeof(double));
  hipMalloc ( &src_vol, tsize * sizeof(double));
  hipMalloc ( &grad, usize * 2 * sizeof(double));
#endif
#ifdef LASRC
  grad_phi_const<<<(usize+BLOCK-1)/BLOCK,BLOCK>>>(grad,usize);
#endif

#ifdef WB
  hipMalloc ( &modes_eq, tsize * sizeof(double));
  hipMalloc ( &edge_eq, tsize*2 * sizeof(double));
  hipMalloc ( &ufaces_pert, 4*nvar*nx*ny*m*sizeof(double));
  hipMalloc ( &w_eq, tsize * sizeof(double));
#endif
  hipMalloc ( &x, nx*ny*m*m * sizeof(double));
  hipMalloc ( &y, nx*ny*m*m * sizeof(double));
  hipMalloc ( &uX, nvar*ny*nx*k*m*sizeof(double));
  hipMalloc ( &uY, nvar*ny*nx*k*m*sizeof(double));
  hipMalloc ( &pivot, tsize * sizeof(double));
  hipMalloc ( &xc, nx*ny * sizeof(double));
  hipMalloc ( &yc, nx*ny * sizeof(double));
  hipMalloc ( &pivot1, tsize * sizeof(double));

  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if(error != hipSuccess){
    printf("Allocated all, CUDA error gpu init: %s\n", hipGetErrorString(error));
    exit(-1);
    mem_check(free,total);
  }
  printf("GPU allocation done\n");
}

extern "C" void gpu_set_pointers_ (double** u_d, double** du_d, double** dudt_d, double** w_d, double** u_eq_d,
			       double** x_d, double** y_d , double** xc_d, double** yc_d, double* x_quad, double* y_quad, double* w_x_quad,
				   double* w_y_quad, double* x_gll, double* y_gll, double* w_x_gll,
				   double* w_y_gll, double* Sqrt_mod) {
  hipError_t error = hipGetLastError();
  *u_d = u;
  *du_d =du;
  *dudt_d =dudt;
  *w_d = w;
  *u_eq_d = u_eq;
  *x_d = x;
  *y_d = y;
  *xc_d = xc;
  *yc_d = yc;
  hipMemcpyToSymbol(HIP_SYMBOL(sqrt_mod),Sqrt_mod,sizeof(double)*m);
  hipMemcpyToSymbol(HIP_SYMBOL(xquad),x_quad,sizeof(double)*m);
  hipMemcpyToSymbol(HIP_SYMBOL(yquad),y_quad,sizeof(double)*m);
  hipMemcpyToSymbol(HIP_SYMBOL(wxquad),w_x_quad,sizeof(double)*m);
  hipMemcpyToSymbol(HIP_SYMBOL(wyquad),w_y_quad,sizeof(double)*m);
  hipMemcpyToSymbol(HIP_SYMBOL(xgll),x_gll,sizeof(double)*k);
  hipMemcpyToSymbol(HIP_SYMBOL(ygll),y_gll,sizeof(double)*k);
  hipMemcpyToSymbol(HIP_SYMBOL(wxgll),w_x_gll,sizeof(double)*k);
  hipMemcpyToSymbol(HIP_SYMBOL(wygll),w_y_gll,sizeof(double)*k);

  hipDeviceSynchronize();
  error = hipGetLastError();
  if(error != hipSuccess){
    printf("CUDA error gpu init: %s\n", hipGetErrorString(error));
    exit(-1);
  }
  printf("FORTRAN-CUDA pointers done\n");
}

extern "C" void gpu_set_more_pointers_ (double* Sqrt_div, double** b_d) {
  hipError_t error = hipGetLastError();
  *b_d = b_modes;
  hipMemcpyToSymbol(HIP_SYMBOL(sqrts_div),Sqrt_div,sizeof(double)*m);

  hipDeviceSynchronize();
  error = hipGetLastError();
  if(error != hipSuccess){
    printf("CUDA error gpu init: %s\n", hipGetErrorString(error));
    exit(-1);
  }
  printf("FORTRAN-CUDA pointers done\n");
}



extern "C" void h2d_ (double *array, double **darray, int* Size) {
  int size = *Size;
  hipError_t error = hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy( *darray, array,  size * sizeof(double) ,hipMemcpyHostToDevice);
  error = hipGetLastError();
  if(error != hipSuccess){
    printf("CUDA error h2d: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

extern "C" void d2h_ (double **darray, double *array, int* Size) {
  int size = *Size;
  hipError_t error = hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy( array, *darray,  size * sizeof(double) ,hipMemcpyDeviceToHost);
  error = hipGetLastError();
  if(error != hipSuccess){
    printf("CUDA error d2h: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}

extern "C" void setdevice_ (int *Device) {
  int device = *Device;
  hipSetDevice(device);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  printf("Device Number: %d\n", device);
  printf("Device name: %s\n", prop.name);
  printf("Device Memory: %lu\n",prop.totalGlobalMem);
}

extern "C" void devices_ () {
  int nDevices;
  hipGetDeviceCount(&nDevices);
  printf("Devices: %d\n",nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Name: %s\n", prop.name);
    printf("  Compute mode: %d\n", prop.computeMode);
    printf("  Memory Capacity (bytes): %lu\n",
           prop.totalGlobalMem);
    printf("  Multiprocessors: %d\n\n", prop.multiProcessorCount);
  }
}
